#include "hip/hip_runtime.h"
#include <complex>
#include <tbb/tbb.h>
#include <fftw3-mpi.h>
# include <cassert>
# include <stdio.h>
# include <time.h>
# include <random>

#include <stdio.h>
#include "fftw.h"
#include <iostream>
#include <fstream>
#include <libconfig.h++>
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "hipfft/hipfft.h"

#include "common.h"
#include "imageReader.h"
#include <ctime>

using std::cout; using std::endl;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

//#define Bits 16
const dim3 threadsPerBlock(16,16);
__device__ __constant__ double cuda_beta_HIO;
__device__ __constant__ int cuda_row;
__device__ __constant__ int cuda_column;
__device__ __constant__ int cuda_rcolor;
__device__ __constant__ double cuda_scale;
__device__ __constant__ int cuda_totalIntensity;
hipfftHandle *plan = new hipfftHandle();
using namespace cv;
double gaussian(double x, double y, double sigma){
  double r2 = pow(x,2) + pow(y,2);
  return exp(-r2/2/pow(sigma,2));
}

double gaussian_norm(double x, double y, double sigma){
  return 1./(2*pi*sigma*sigma)*gaussian(x,y,sigma);
}

/******************************************************************************/

void maskOperation(Mat &input, Mat &output, Mat &kernel){
  filter2D(input, output, input.depth(), kernel);
}

class support{
public:
  support(){};
  __device__ __host__ virtual bool isInside(int x, int y) = 0;
};
class ImageMask{
public:
  int nrow;
  int ncol;
  size_t sz;
  double *data;
  Mat *image;
  double threshold;
  ImageMask *cuda;
  ImageMask(){
    hipMalloc(&cuda, sizeof(ImageMask));
  };
  void init_image(Mat* image_){
    nrow = image_->rows;
    ncol = image_->cols;
    image = image_;
    sz = image_->total()*sizeof(double);
    hipMalloc((void**)&data,sz);
  }
  void updateCuda(){
    hipMemcpy(cuda, this, sizeof(ImageMask), hipMemcpyHostToDevice);
  }
  void cpyToGM(){
    hipMemcpy(data, image->data, sz, hipMemcpyHostToDevice);
  }
  void cpyFromGM(){
    hipMemcpy(image->data, data, sz, hipMemcpyDeviceToHost);
  }
  __device__ __host__ bool isInside(int x, int y){
    if(data[x*ncol+y] < threshold) {
	    //printf("%d, %d = %f lower than threshold, dropping\n",x,y,image->ptr<double>(x)[y]);
	    return false;
    }
    return true;
  }
};
class rect{
public:
  int startx;
  int starty;
  int endx;
  int endy;
  __device__ __host__ bool isInside(int x, int y){
    if(x > startx && x <= endx && y > starty && y <= endy) return true;
    return false;
  }
};
class C_circle{
public:
  int x0;
  int y0;
  double r;
  __device__ __host__ bool isInside(int x, int y){
    double dr = sqrt(pow(x-x0,2)+pow(y-y0,2));
    if(dr < r) return true;
    return false;
  }
};

template <typename sptType>
__global__ void createMask(double* data, sptType* spt, bool isFrequency=0){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  if(isFrequency){
    if(x>=cuda_row/2) x-=cuda_row/2;
    else x+=cuda_row/2;
    if(y>=cuda_column/2) y-=cuda_column/2;
    else y+=cuda_column/2;
  }
  data[index]=spt->isInside(x,y);
}

Mat* gaussianKernel(int rows, int cols, double sigma){
  Mat* image = new Mat(rows, cols, CV_64FC1);
  auto f = [&](int x, int y, double &data){
    data = gaussian_norm(x-rows/2,y-cols/2,sigma);
  };
  imageLoop<decltype(f), double>(image,&f);
  
  return image;
}
/******************************************************************************/
__global__ void applyNorm(hipfftDoubleComplex* data){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  data[index].x*=1./sqrtf(cuda_row*cuda_column);
  data[index].y*=1./sqrtf(cuda_row*cuda_column);
}

__global__ void applyMod(hipfftDoubleComplex* source, hipfftDoubleComplex* target, ImageMask *bs = 0, bool loose=0){
  assert(source!=0);
  assert(target!=0);
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  if(loose && bs && bs->isInside(x,y)) return;
  double tolerance = 0.5/cuda_rcolor*cuda_scale*16+30./cuda_rcolor; // fluctuation caused by bit depth and noise
  double maximum = pow(mergeDepth,2)*cuda_scale*0.99;
  int index = x*cuda_column + y;
  hipfftDoubleComplex targetdata = target[index];
  hipfftDoubleComplex sourcedata = source[index];
  double ratiox = 1;
  double ratioy = 1;
  double mod2 = targetdata.x*targetdata.x + targetdata.y*targetdata.y;
  double srcmod2 = sourcedata.x*sourcedata.x + sourcedata.y*sourcedata.y;
  if(mod2>=maximum) {
    if(loose) mod2 = max(maximum,srcmod2);
    else tolerance*=1000;
  }
  double diff = mod2-srcmod2;
  if(diff>tolerance){
    ratioy=ratiox = sqrt((mod2-tolerance)/srcmod2);
  }else if(diff < -tolerance ){
    ratioy=ratiox = sqrt((mod2+tolerance)/srcmod2);
  }
  if(srcmod2 == 0){
    ratiox = sqrt(mod2);
    ratioy = 0;
  }
  source[index].x = ratiox*sourcedata.x;
  source[index].y = ratioy*sourcedata.y;
}

Mat* createWaveFront(Mat &intensity, Mat &phase, int rows, int columns, Mat* &itptr, Mat* wavefront = 0){
  if ( !intensity.data )
  {
      printf("No intensity data \n");
      exit(0);
  }
  if ( !phase.data )
  {
      printf("No phase data \n");
      exit(0);
  }
  if(intensity.rows!=phase.rows || intensity.cols!=phase.cols) {
    printf("intensity map and phase map having different dimensions");
    exit(0);
  }
  columns = intensity.cols;
  rows = intensity.rows;
  Mat *imageptr;
  itptr = &intensity;
  Mat &intensity_sc = *itptr;
  if(phase.channels()==3){
    imageptr = new Mat(rows, columns, format_cv);
    cv::cvtColor(phase, *imageptr, cv::COLOR_BGR2GRAY);
  }else{
    imageptr = &phase;
  }
  Mat &phase_sc = *imageptr;
  //wavefront = convertFromIntegerToComplex(intensity_sc, wavefront,0,"waveFront");
  wavefront = convertFromIntegerToComplex(intensity_sc, phase_sc, wavefront);
  delete imageptr;
  return wavefront;
  //imwrite("input.png",image);
}


void ApplyERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0];
    rhonp1[1] = rhoprime[1];
  }else{
    rhonp1[0] = rhonp1[1] = 0;
  }
}
void ApplyPOSERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0]*( rhoprime[0] > 0 );
    rhonp1[1] = 0;
  }else{
    rhonp1[0] = rhonp1[1] = 0;
  }
}
void ApplyLoosePOSERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime, double threshold){
  if(rhoprime[0] < threshold){
    rhonp1[0] = rhoprime[0]*( rhoprime[0] > 0 );
  }else{
    rhonp1[0] = threshold;
  }
    rhonp1[1] = 0;
}
__device__ void ApplyHIOSupport(bool insideS, hipfftDoubleComplex &rhonp1, hipfftDoubleComplex &rhoprime, double beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
    rhonp1.y -= beta*rhoprime.y;
  }
}
__device__ void ApplyPOSHIOSupport(bool insideS, hipfftDoubleComplex &rhonp1, hipfftDoubleComplex &rhoprime, double beta){
  if(rhoprime.x > 0 && (insideS/* || rhoprime[0]<30./rcolor*/)){
    rhonp1.x = rhoprime.x;
  }else{
    rhonp1.x -= beta*rhoprime.x;
  }
  rhonp1.y -= beta*rhoprime.y;
}
void ApplyLoosePOSHIOSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime, double beta, double threshold){
  if(rhoprime[0] > 0 && (rhoprime[0]<threshold)){
    rhonp1[0] = rhoprime[0];
    //rhonp1[1] = rhoprime[1];
    rhonp1[1] -= beta*rhoprime[1];
  }else{
    rhonp1[0] -= beta*(rhoprime[0]);
    rhonp1[1] -= beta*rhoprime[1];
  }
}
void ApplySFSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0];
    rhonp1[1] = rhoprime[1];
  }else{
    rhonp1[0] = -0.9*rhoprime[0];
    rhonp1[1] = -0.9*rhoprime[1];
  }
}
void ApplyDMSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhop, fftw_complex &pmsrho, double gammas, double gammam, double beta){

  complex<double> &rho = *(complex<double>*)rhonp1;
  complex<double> &rhoprime = *(complex<double>*)rhop;
  complex<double> &pmpsrho = *(complex<double>*)pmsrho;
  if(1||insideS){
    rho = 2.*pmpsrho-rhoprime;//(1-beta*gammam)*rhoprime+beta*(1+gammam+gammas)*rhoprime-beta*(1+gammas)*pmpsrho;
  }else{
    rho += 2.*pmpsrho-rhoprime;//beta*gammas*rhoprime-beta*(1+gammas)*pmpsrho;
  }
}
void ApplyPOSDMSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhop, fftw_complex &pmsrho, double gammas, double gammam, double beta){

  complex<double> rho(rhonp1[0],rhonp1[1]);
  complex<double> rhoprime(rhop[0],rhop[1]);
  complex<double> pmpsrho(pmsrho[0],pmsrho[1]);
  if(insideS){
    rho = (1-beta*gammam)*rhoprime+beta*(1+gammam+gammas)*rhoprime-beta*(1+gammas)*pmpsrho;
  }else{
    rho += beta*gammas*rhoprime-beta*(1+gammas)*pmpsrho;
  }
  rhonp1[0] = rho.real();
  if(rhonp1[0]<0) rhonp1[0] = 0;
  rhonp1[1] = 0;
}
struct experimentConfig{
 bool useDM;
 bool useBS;
 bool useShrinkMap = 0;
 bool reconAC = 0;
 ImageMask* spt;
 ImageMask* beamStop;
 bool restart;
 double lambda = 0.6;
 double d = 16e3;
 double pixelsize = 6.5;
 double beamspotsize = 50;
};

__global__ void applySupport(hipfftDoubleComplex *gkp1, hipfftDoubleComplex *gkprime, double* objMod, ImageMask *spt){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = x*cuda_column + y;
  if(x >= cuda_row || y >= cuda_column) return;

  //epsilonF+=hypot(gkp1data[0]-gkprimedata[0],gkp1data[1]-gkprimedata[1]);
  //fftw_complex tmp = {gkp1data[0],gkp1data[1]};
  bool inside = spt->isInside(x,y);
  hipfftDoubleComplex &gkp1data = gkp1[index];
  hipfftDoubleComplex &gkprimedata = gkprime[index];
  //if(iter >= niters - 20 ) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20 || iter % 200 == 0) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20 || iter<20) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20) ApplyERSupport(inside,gkp1data,gkprimedata);
  //ApplyERSupport(inside,gkp1data,gkprimedata);
  //else ApplyHIOSupport(inside,gkp1data,gkprimedata,beta_HIO);
  //else ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,beta_HIO);
  //printf("%d, (%f,%f), (%f,%f), %f\n",inside, gkprimedata.x,gkprimedata.y,gkp1data.x,gkp1data.y,cuda_beta_HIO);
  ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  objMod[index] = hipCabs(gkp1data);
  //double thres = gaussian(x-row/2,y-column/2,40);
  //ApplyLoosePOSHIOSupport(inside,gkp1data,gkprimedata,beta_HIO,thres);
  //ApplyLoosePOSERSupport(inside,gkp1data,gkprimedata,thres);
  //else {
  //ApplyDMSupport(inside,gkp1data, gkprimedata, pmpsg[index], gammas, gammam, beta);
  //}
  //ApplyERSupport(inside,pmpsg[index],gkp1data);
  //ApplyHIOSupport(inside,gkp1data,gkprimedata,beta);
  //else ApplySFSupport(inside,gkp1data,gkprimedata);
  //epsilonS+=hypot(tmp[0]-gkp1data[0],tmp[1]-gkp1data[1]);
}
void phaseRetrieve( experimentConfig &setups, Mat* targetfft, Mat* gkp1 = 0, Mat *cache = 0, Mat* fftresult = 0 ){
    Mat* pmpsg = 0;
    bool useShrinkMap = setups.useShrinkMap;
    int row = targetfft->rows;
    int column = targetfft->cols;
    bool useDM = setups.useDM;
    bool useBS = setups.useBS;
    ImageMask &re = *setups.spt;
    auto &cir = *(setups.beamStop);
    if(useDM) {
      pmpsg = new Mat();
      fftresult->copyTo(*pmpsg);
    }
    if(gkp1==0) gkp1 = new Mat(row,column,CV_64FC2);
    assert(targetfft!=0);
    double beta = -1;
    double beta_HIO = 0.9;
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_beta_HIO),&beta_HIO,sizeof(beta_HIO));
    double gammas = -1./beta;
    double gammam = 1./beta;
    double epsilonS, epsilonF;
    std::ofstream fepF,fepS;
    fepF.open("epsilonF.txt",ios::out |(setups.restart? ios::app:std::ios_base::openmode(0)));
    fepS.open("epsilonS.txt",ios::out |(setups.restart? ios::app:std::ios_base::openmode(0)));
    int niters = 5000;
    int tot = row*column;
    bool saveIter=1;
    Mat objMod(row,column,CV_64FC1);
    Mat* maskKernel;
    double gaussianSigma = 3;

    size_t sz = row*column*sizeof(hipfftDoubleComplex);
    hipfftDoubleComplex *cuda_fftresult, *cuda_targetfft, *cuda_gkprime, *cuda_gkp1, *cuda_pmpsg;
    double *cuda_objMod;
    ImageMask *cuda_spt;
    hipMalloc((void**)&cuda_fftresult, sz);
    hipMalloc((void**)&cuda_targetfft, sz);
    hipMalloc((void**)&cuda_gkprime, sz);
    hipMalloc((void**)&cuda_gkp1, sz);
    hipMalloc((void**)&cuda_objMod, sz/2);
    hipMalloc((void**)&cuda_spt, sizeof(ImageMask));
    hipMemcpy(cuda_spt, &re, sizeof(ImageMask), hipMemcpyHostToDevice);
    hipMemcpy(cuda_targetfft, targetfft->data, sz, hipMemcpyHostToDevice);
    hipMemcpy(cuda_fftresult, fftresult->data, sz, hipMemcpyHostToDevice);

    dim3 numBlocks((row-1)/threadsPerBlock.x+1, (column-1)/threadsPerBlock.y+1);
    //dim3 numBlocks(row/threadsPerBlock.x, column/threadsPerBlock.y);

    hipfftExecZ2Z( *plan, cuda_targetfft, cuda_gkp1, HIPFFT_BACKWARD);
    applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_gkp1);
    hipDeviceSynchronize();
    std::chrono::time_point<std::chrono::high_resolution_clock> now = std::chrono::high_resolution_clock::now();
    std::chrono::duration<int64_t, std::nano> time_applyMod(0);
    std::chrono::duration<int64_t, std::nano> time_FFT(0);
    std::chrono::duration<int64_t, std::nano> time_support(0);
    std::chrono::duration<int64_t, std::nano> time_norm(0);
    for(int iter = 0; iter < niters; iter++){
      //start iteration
      if(iter%100==0) {
	long tot = time_FFT.count()+time_norm.count()+time_support.count()+time_applyMod.count();
	if(tot!=0)
	printf("iter: %d, timing:\n  FFT:%ld, %4.2f%%\n  NORM:%ld, %4.2f%%\n  Support:%ld, %4.2f%%\n  applyMod:%ld, %4.2f%%\n",iter, 
	  time_FFT.count(),     ((double)time_FFT.count())/tot*100,
	  time_norm.count(),    ((double)time_norm.count())/tot*100,
	  time_support.count(), ((double)time_support.count())/tot*100,
	  time_applyMod.count(),((double)time_applyMod.count())/tot*100
        );
	if(saveIter){
          hipMemcpy(gkp1->data, cuda_gkp1, sz, hipMemcpyDeviceToHost);
          convertFromComplexToInteger( gkp1,cache, MOD2,0);
          std::string iterstr = to_string(iter);
          imwrite("recon_intensity"+iterstr+".png",*cache);
          convertFromComplexToInteger( gkp1,cache, PHASE,0);
          imwrite("recon_phase"+iterstr+".png",*cache);
	}
      }
      now = std::chrono::high_resolution_clock::now();
      if(useBS) applyMod<<<numBlocks,threadsPerBlock>>>(cuda_fftresult,cuda_targetfft,cir.cuda, !setups.reconAC || iter > 1000);  //apply mod to fftresult, Pm
      else applyMod<<<numBlocks,threadsPerBlock>>>(cuda_fftresult,cuda_targetfft,0, !setups.reconAC || iter > 1000);  //apply mod to fftresult, Pm
      if(useDM) {
        if(useBS) applyMod<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg,cuda_targetfft,cir.cuda, !setups.reconAC || iter > 1000);  
        else applyMod<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg,cuda_targetfft,0, !setups.reconAC || iter > 1000);
      }
      //hipDeviceSynchronize();
      time_applyMod+=std::chrono::high_resolution_clock::now()-now;
      
      epsilonS = epsilonF = 0;
      now = std::chrono::high_resolution_clock::now();
      hipfftExecZ2Z( *plan, cuda_fftresult, cuda_gkprime, HIPFFT_BACKWARD);
      time_FFT+=std::chrono::high_resolution_clock::now()-now;
      now = std::chrono::high_resolution_clock::now();
      applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_gkprime);
     // hipDeviceSynchronize();
      time_norm+=std::chrono::high_resolution_clock::now()-now;
      if(useDM){
        now = std::chrono::high_resolution_clock::now();
        hipfftExecZ2Z( *plan, cuda_pmpsg, cuda_pmpsg, HIPFFT_BACKWARD);
        time_FFT+=std::chrono::high_resolution_clock::now()-now;
        now = std::chrono::high_resolution_clock::now();
        applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg);
        hipDeviceSynchronize();
        time_norm+=std::chrono::high_resolution_clock::now()-now;
      }
      bool updateMask = (iter > 1000) && iter%20==0 && useShrinkMap && iter!=0;
      if(updateMask){
        int size = floor(gaussianSigma*6); // r=3 sigma to ensure the contribution is negligible (0.01 of the maximum)
        size = size/2*2+1; //ensure odd
        maskKernel = gaussianKernel(size,size,gaussianSigma);
      }
      now = std::chrono::high_resolution_clock::now();
      applySupport<<<numBlocks,threadsPerBlock>>>(cuda_gkp1, cuda_gkprime, cuda_objMod,cuda_spt);
      time_support+=std::chrono::high_resolution_clock::now()-now;
      /*
        hipMemcpy(gkp1->data, cuda_gkp1, sz, hipMemcpyDeviceToHost);
        convertFromComplexToInteger( gkp1,cache, MOD2,0);
        imwrite("debug.png",*cache);
	*/
      //hipDeviceSynchronize();
      if(updateMask){
        hipMemcpy(objMod.data, cuda_objMod, sz/2, hipMemcpyDeviceToHost);
        filter2D(objMod, *re.image,objMod.depth(),*maskKernel);
	((ImageMask*)&re)->cpyToGM();
	if(gaussianSigma>1.5) gaussianSigma*=0.99;
	delete maskKernel;
      }
      if(updateMask&&iter%100==0&&saveIter){
	convertFromComplexToInteger<double>(re.image, cache,MOD,0);
        std::string iterstr = to_string(iter);
	imwrite("mask"+iterstr+".png",*cache);
      }
      if(iter!=0){
        fepF<<sqrt(epsilonF/tot)<<endl;
        fepS<<sqrt(epsilonS/tot)<<endl;
      }else {
        hipMemcpy(gkp1->data, gkp1, sz, hipMemcpyDeviceToHost);
        convertFromComplexToInteger( gkp1,cache, MOD2,0);
        imwrite("recon_support.png",*cache);
        convertFromComplexToInteger( gkp1,cache, PHASE,0);
        imwrite("recon_phase_support.png",*cache);
      }

      //if(sqrt(epsilonS/row/column)<0.05) break;
      now = std::chrono::high_resolution_clock::now();
      hipfftExecZ2Z( *plan, cuda_gkp1, cuda_fftresult, HIPFFT_FORWARD);
      time_FFT+=std::chrono::high_resolution_clock::now()-now;
      now = std::chrono::high_resolution_clock::now();
      applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_fftresult);
      //hipDeviceSynchronize();
      time_norm+=std::chrono::high_resolution_clock::now()-now;
      if(useDM){ // FFT to get f field;
        hipfftExecZ2Z( *plan, cuda_pmpsg, cuda_pmpsg, HIPFFT_FORWARD);
        applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg);
        hipDeviceSynchronize();
      }
      //end iteration
    }
    fepF.close();
    fepS.close();
    hipMemcpy(fftresult->data, cuda_fftresult, sz, hipMemcpyDeviceToHost);
    hipMemcpy(targetfft->data, cuda_targetfft, sz, hipMemcpyDeviceToHost);
    hipMemcpy(gkp1->data, gkp1, sz, hipMemcpyDeviceToHost);

    convertFromComplexToInteger( gkp1,cache, MOD2,0);
    imwrite("recon_intensity.png",*cache);
    convertFromComplexToInteger(gkp1, cache, PHASE,0);
    imwrite("recon_phase.png",*cache);
    if(useDM)  convertFromComplexToInteger( pmpsg, cache, MOD2,1);
    if(useDM)  imwrite("recon_pmpsg.png",*cache);
    convertFromComplexToInteger( fftresult, cache, MOD2,1);
    imwrite("recon_pattern.png",*cache);
}

__global__ void applyAutoCorrelationMod(hipfftDoubleComplex* source,hipfftDoubleComplex* target, ImageMask *bs = 0){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  double targetdata = target[index].x;
  double retval = targetdata;
  source[index].y = 0;
  double maximum = pow(mergeDepth,2)*cuda_scale*0.99;
  double sourcedata = source[index].x;
  double tolerance = 0.5/cuda_rcolor*cuda_scale;
  double diff = sourcedata-targetdata;
  if(bs && bs->isInside(x,y)) {
    if(targetdata<0) target[index].x = 0;
    return;
  }
  if(diff>tolerance){
    retval = targetdata+tolerance;
  }else if(diff < -tolerance ){
    retval = targetdata-tolerance;
  }else{
    retval = targetdata;
  }
  if(targetdata>=maximum) {
    retval = max(sourcedata,maximum);
  }
  source[index].x = retval;
}

template <typename sptType>
__global__ void applyERACSupport(hipfftDoubleComplex* data,hipfftDoubleComplex* prime,sptType *spt, double* objMod){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  if(!spt->isInside(x,y)){
    data[index].x = 0;
    data[index].y = 0;
  }
  else{
    data[index].x = prime[index].x;
    data[index].y = prime[index].y;
  }
  objMod[index] = hipCabs(data[index]);
}

template <typename sptType>
__global__ void applyHIOACSupport(hipfftDoubleComplex* data,hipfftDoubleComplex* prime, sptType *spt, double *objMod){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  if(!spt->isInside(x,y)){
    data[index].x -= prime[index].x;
  }
  else{
    data[index].x = prime[index].x;
  }
  data[index].y -= prime[index].y;
  objMod[index] = hipCabs(data[index]);
}

template <typename sptType>
void autoCorrelationConstrain(Mat* pattern, sptType *spt, Mat* cache, ImageMask *bs = 0){  //beam stop
  double totalIntensity = 1000;
  hipMemcpyToSymbol(HIP_SYMBOL(cuda_totalIntensity),&totalIntensity,sizeof(totalIntensity));
  dim3 numBlocks((pattern->rows-1)/threadsPerBlock.x+1, (pattern->cols-1)/threadsPerBlock.y+1);
  hipfftDoubleComplex *autocorrelation, *cuda_pattern, *autoprime;
  ImageMask autoCorrelationMask;
  ImageMask *cuda_Mask;
  autoCorrelationMask.init_image(new Mat(pattern->rows,pattern->cols,CV_64FC1));
  autoCorrelationMask.threshold = 0.008;
  sptType *cuda_spt;
  size_t sz = pattern->total()*sizeof(hipfftDoubleComplex);
  double *cuda_objMod;
  hipMalloc((void**)&cuda_objMod, sz/2);
  hipMalloc((void**)&autocorrelation,sz);
  hipMalloc((void**)&autoprime,sz);
  hipMalloc((void**)&cuda_pattern,sz);
  hipMalloc((void**)&cuda_spt,sizeof(sptType));
  hipMalloc((void**)&cuda_Mask,sizeof(ImageMask));
  hipMemcpy(cuda_pattern, pattern->data, sz, hipMemcpyHostToDevice);
  hipMemcpy(cuda_spt, spt, sizeof(sptType), hipMemcpyHostToDevice);
  hipMemcpy(cuda_Mask, &autoCorrelationMask, sizeof(ImageMask), hipMemcpyHostToDevice);
  createMask<sptType><<<numBlocks,threadsPerBlock>>>(autoCorrelationMask.data, cuda_spt,1);

  autoCorrelationMask.cpyFromGM();
  convertFromComplexToInteger<double>(autoCorrelationMask.image, cache,MOD, 1);
  imwrite("autoCorrelationMask.png",*cache);
  hipfftExecZ2Z(*plan,cuda_pattern,autocorrelation,HIPFFT_BACKWARD);
  applyNorm<<<numBlocks,threadsPerBlock>>>(autocorrelation);
  hipMemcpy(pattern->data, autocorrelation, sz, hipMemcpyDeviceToHost);
  convertFromComplexToInteger(pattern, cache, REAL, 1);
  imwrite("initAC.png",*cache);
  double gaussianSigma=3;
  Mat* maskKernel;
  for(int iter = 0; iter < 5000; iter++){
    bool updateMask = iter%20==0 && iter>2000;
    if(iter%100==0){
      printf("auto correlation iteration: %d\n",iter);
      hipMemcpy(pattern->data, autoprime, sz, hipMemcpyDeviceToHost);
      convertFromComplexToInteger(pattern, cache, REAL, 1);
      imwrite("accorrected"+to_string(iter)+".png",*cache);
    }
    if(iter<4500) applyHIOACSupport<<<numBlocks,threadsPerBlock>>>(autocorrelation, autoprime, cuda_Mask, cuda_objMod);
    else applyERACSupport<<<numBlocks,threadsPerBlock>>>(autocorrelation, autoprime, cuda_Mask, cuda_objMod);
    hipfftExecZ2Z(*plan,autocorrelation,autoprime,HIPFFT_FORWARD);
    applyNorm<<<numBlocks,threadsPerBlock>>>(autoprime);
    applyAutoCorrelationMod<<<numBlocks,threadsPerBlock>>>(autoprime, cuda_pattern, bs?bs->cuda:0);
    hipfftExecZ2Z(*plan,autoprime,autoprime,HIPFFT_BACKWARD);
    applyNorm<<<numBlocks,threadsPerBlock>>>(autoprime);
    if(updateMask){
      int size = floor(gaussianSigma*6); // r=3 sigma to ensure the contribution is negligible (0.01 of the maximum)
      size = size/2*2+1; //ensure odd
      maskKernel = gaussianKernel(size,size,gaussianSigma);
      hipMemcpy(autoCorrelationMask.image->data, cuda_objMod, sz/2, hipMemcpyDeviceToHost);
      filter2D(*autoCorrelationMask.image, *autoCorrelationMask.image,autoCorrelationMask.image->depth(),*maskKernel);
      autoCorrelationMask.cpyToGM();
      if(gaussianSigma>1.5) gaussianSigma*=0.99;
      delete maskKernel;
    }
  }
  hipFree(cuda_pattern);
  hipFree(cuda_spt);
  hipFree(cuda_Mask);
  hipFree(autocorrelation);
  hipMemcpy(pattern->data, autoprime, sz, hipMemcpyDeviceToHost);
  hipFree(autoprime);
  convertFromComplexToInteger(pattern, cache, REAL, 1);
  imwrite("accorrected.png",*cache);
}

int main(int argc, char** argv )
{

    if(argc < 2){
      printf("please feed the object intensity and phase image\n");
    }
    bool runSim;
    bool simCCDbit = 1;
    printf("command:");
    for(int i = 0; i < argc ; i++){
	    printf("%s ",argv[i]);
    }
    printf("\n");
    if(argv[1] == std::string("sim")){
      runSim = 1;
    }else{
      runSim = 0;
    }


    auto seed = (unsigned)time(NULL);
    bool isFresnel = 0;
    bool doIteration = 1;
    bool useGaussionLumination = 0;
    bool useGaussionHERALDO = 0;
    bool useRectHERALDO = 0;

    //int seed = 1657180926;// 1657180330
    //int seed = 1657182238; // oversampling = 3, modulation range = pi, reversed image
    //1657182741 // oversampling = 3, modulation range = 1.1pi, reversed image
    //1657182948 // oversampling = 3, modulation range = 1.3pi, reversed image
    //1657184141 // oversampling = 3, modulation range = 2pi, upright image, random phase
    srand(seed);
    printf("seed:%d\n",seed);
    double oversampling = 4;
    Mat* gkp1 = 0;
    Mat* targetfft = 0;
    Mat* fftresult = 0;
    bool restart = 0;
    if(argc > 4){
      restart = 1;
      
    }
    int row, column;
    Mat intensity = readImage( argv[2] , !runSim);
    //maskOperation(intensity,intensity);
    Mat ele = getStructuringElement(MORPH_RECT,Size(3,3),Point(1,1));
    //erode( intensity, intensity, ele);
    //dilate( intensity, intensity, ele);
    row = intensity.rows;
    column = intensity.cols;
    pixeltype *rowp;
    if(useRectHERALDO){
      for(int i = 0; i < row ; i++){
        rowp = intensity.ptr<pixeltype>(i);
        for(int j = 0; j < column ; j++){
          if(i > row/2 || j > column/2) rowp[j] = rcolor-1;
	}
      }
    }


    if(runSim){
	    row*=oversampling;
	    column*=oversampling;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_row),&row,sizeof(row));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_column),&column,sizeof(column));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_rcolor),&rcolor,sizeof(rcolor));
    //double tmp = scale*10;
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_scale),&scale,sizeof(scale));

    hipfftPlan2d ( plan, row, column, HIPFFT_Z2Z);

    C_circle cir,cir2,cir3;
    //cir is the beam stop
    //cir.x0=row/2-50;
    //cir.y0=column/2+20;
    //cir.r=50;
    cir.x0=row/2;
    cir.y0=column/2;
    cir.r=10;
    //cir2.x0 = column*2/3-50;
    //cir2.y0 = row*2/3+110;
    //cir2.r = 50;
    cir2.x0 = column/2;
    cir2.y0 = row/2;
    cir2.r = 40;
    cir3.x0 = row/2;
    cir3.y0 = column/2;
    //cir3.r = 300/mergeDepth;
    cir3.r = 40;
    rect re;
    re.startx = (oversampling-1)/2*row/oversampling;
    re.starty = (oversampling-1)/2*column/oversampling;
    //re.startx = 1./4*row;
    //re.starty = 1./4*column;
    re.endx = row-re.startx;
    re.endy = column-re.starty;
    
    dim3 numBlocks((row-1)/threadsPerBlock.x+1, (column-1)/threadsPerBlock.y+1);

    experimentConfig setups;
    setups.reconAC = 0;
    setups.useShrinkMap = 1;
    ImageMask shrinkingMask;
    shrinkingMask.threshold = 0.1;
    setups.useDM = 0;

    setups.useBS = 1;
    ImageMask beamStop;
    beamStop.threshold = 0.5;
    beamStop.init_image(new Mat(row,column,CV_64FC1));
    C_circle *cuda_spt;
    hipMalloc((void**)&cuda_spt,sizeof(C_circle));
    hipMemcpy(cuda_spt, &cir, sizeof(C_circle), hipMemcpyHostToDevice);
    createMask<<<numBlocks,threadsPerBlock>>>(beamStop.data, cuda_spt,1);
    beamStop.cpyFromGM();
    hipFree(cuda_spt);
    beamStop.updateCuda();

    setups.spt = &shrinkingMask;
    //setups.spt = &re;
    //setups.spt = &cir3;
    
    setups.beamStop = &beamStop;//&cir;
    setups.restart = restart;
    //setups.d = oversampling*setups.pixelsize*setups.beamspotsize/setups.lambda; //distance to guarentee oversampling
    setups.pixelsize = 7;//setups.d/oversampling/setups.beamspotsize*setups.lambda;
    printf("recommanded imaging distance = %f\n", setups.d);
    printf("recommanded pixel size = %f\n", setups.pixelsize);

    bool isFarField = 0;
    double reversefresnelNumber = setups.d*setups.lambda/pi/pow(setups.beamspotsize,2);
    printf("Fresnel Number = %f\n",1./reversefresnelNumber);
    if(reversefresnelNumber > 100) isFarField = 1;
    //these are for simulation
    Mat* cache = 0;
    Mat* cache1;
    cache1 = &intensity;
    if(runSim){
      if(argc==4){
        Mat phase = readImage( argv[3]);
        if(oversampling>1) 
          gkp1 = createWaveFront(*extend(*cache1,oversampling), *extend(phase,oversampling), row, column,cache,gkp1);
        //if(oversampling>1) gkp1 = createWaveFront(extend(intensity,oversampling), extend(phase,oversampling), row, column,cache,gkp1);
        else gkp1 = createWaveFront(intensity,phase, row, column,cache,gkp1);
      }else{
        if(oversampling>1) cache = extend(*cache1,oversampling);
	else cache = cache1;
	gkp1 = convertFromIntegerToComplex(*cache, gkp1,0,"waveFront");
      }
      if(!isFarField && isFresnel){
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
	  double phase = pi*setups.lambda*setups.d/pow(setups.pixelsize,2)*(pow((x-0.5*row)/row,2)+pow((y-0.5*column)/column,2))/10;
	  *tmp *= exp(complex<double>(0,phase));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      if(useGaussionLumination){
        //setups.spt = &re;
        //if(!setups.useShrinkMap) setups.spt = &cir3;
        //diffraction image, either from simulation or from experiments.
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
          bool inside = cir3.isInside(x,y);
	  if(!inside) *tmp = 0.;
	  *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir3.r);
	  //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      if(useGaussionHERALDO){
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
	  if(cir2.isInside(x,y)) 
		  *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
	  else *tmp = gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
	  if(x < row*1/3 && y < row*1/3) *tmp = 0;
	  //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      convertFromComplexToInteger(gkp1, cache, MOD2,0,1,"Object MOD2");
      imwrite("init_object.png",*cache);
      convertFromComplexToInteger(gkp1, cache, PHASE,0,1,"Object Phase");
      imwrite("init_object_phase.png",*cache);
      targetfft = fftw(gkp1,targetfft,1); 
    }else{
      //if(mergeDepth == 1) cache = cache1;
      //else 
      cache = new Mat(row, column, format_cv);
      if(cache1->depth() == CV_64F) 
        targetfft = cache1;
      else
        targetfft = convertFromIntegerToComplex(*cache1,targetfft,1); 
    }
    if(restart){
      intensity = readImage(argv[3]);
      Mat phase = readImage(argv[4]);
      gkp1 = createWaveFront(intensity, phase, row, column,cache,gkp1);
      fftresult = fftw(gkp1,fftresult,1); //If not restart, this line just allocate space, the values are not used.
    }
    //cir2.x0=row/2;
    //cir2.y0=column/2;
    double decay = scale;
    if(runSim) decay=1;
    std::default_random_engine generator;
    double noiseLevel = 9;
    std::poisson_distribution<int> distribution(noiseLevel);
    Mat *autocorrelation = new Mat(row,column,CV_64FC2,Scalar::all(0.));
    shrinkingMask.init_image(new Mat(row,column,CV_64FC1));
    for(int i = 0; i<row*column; i++){ //remove the phase information
     // double randphase = arg(tmp);//static_cast<double>(rand())/RAND_MAX*2*pi;
      complex<double> &data = *(complex<double>*)((fftw_complex*)targetfft->data)[i];
      fftw_complex &datacor = ((fftw_complex*)autocorrelation->data)[i];
      double mod = abs(data)*sqrt(decay);
      if(runSim&&simCCDbit) {
        int range= pow(2,12);
        mod = sqrt(((double)floor(pow(mod,2)*range))/(range)); //assuming we use 16bit CCD
        mod = sqrt(max(0.,pow(mod,2)+double(distribution(generator)-noiseLevel)/range)); //Poisson noise
      }
      if(1){
        if(setups.useBS && ((double*)beamStop.image->data)[i]>0.5) {
          data = 0.;
	  mod = 0;
        }
        else{
          //complex<double> tmp(targetfft[i][0],targetfft[i][1]);
          double randphase = static_cast<double>(rand())/RAND_MAX*2*pi;
          data = mod*exp(complex<double>(0,randphase));
        }
      }
      //datacor[0] = pow(mod,2)*(tx-row/2)*(ty-column/2)/90; // ucore is the derivitaves of the diffraction pattern: append *(tx-row/2)*(ty-column/2)/20;
      datacor[0] = pow(mod,2); //ucore is the diffraction pattern
      datacor[1] = 0;
    }
    convertFromComplexToInteger( autocorrelation, cache, REAL,1,1,"HERALDO U core"); 
    imwrite("ucore.png",*cache);
    if(!setups.reconAC || runSim) {
      autocorrelation = fftw(autocorrelation, autocorrelation, 0);
      rect *cuda_spt;
      hipMalloc((void**)&cuda_spt,sizeof(rect));
      hipMemcpy(cuda_spt, &re, sizeof(rect), hipMemcpyHostToDevice);
      createMask<<<numBlocks,threadsPerBlock>>>(shrinkingMask.data, cuda_spt,0);
      shrinkingMask.cpyFromGM();
      hipFree(cuda_spt);
      shrinkingMask.updateCuda();
    }
    else {
      autoCorrelationConstrain(autocorrelation, &re, cache, setups.useBS?&beamStop:0);
      fftw(autocorrelation, targetfft, 1);
      auto f =  [&](int x, int y, fftw_complex &data){
	data[0] = sqrt(max(data[0],0.));
	data[1] = 0;
      };
      imageLoop<decltype(f)>(targetfft, &f, 0);
      auto f1 = [&](int x, int y, double &data, fftw_complex &dataout){
        data = hypot(dataout[1],dataout[0])>shrinkingMask.threshold;
      };
      imageLoop<decltype(f1),double,fftw_complex>(shrinkingMask.image,autocorrelation,&f1,1);
      shrinkingMask.cpyToGM();
    }

    convertFromComplexToInteger<double>(shrinkingMask.image, cache,MOD,0);
    imwrite("mask.png",*cache);
    //auto f = [&](int x, int y, fftw_complex &data){
    //  auto tmp = (complex<double>*)&data;
    //  *tmp = 1.4+*tmp;
    //};
    //imageLoop<decltype(f)>(autocorrelation,&f,0);
    if(!restart){
      fftresult = new Mat();
      targetfft->copyTo(*fftresult);
    }
    convertFromComplexToInteger(targetfft, cache, PHASE,1);
    imwrite("init_phase.png",*cache);
    convertFromComplexToInteger(targetfft, cache, MOD2,1,1,"Pattern MOD2");
    imwrite("init_pattern.png",*cache);
    //if(runSim) targetfft = convertFromIntegerToComplex(*cache, targetfft, 1 , "waveFront");
    convertFromComplexToInteger(autocorrelation, cache, REAL,1,1,"Autocorrelation MOD2",1);
    imwrite("auto_correlation.png",*cache);
    //Phase retrieving starts from here. In the following, only targetfft is needed.
    if(doIteration) phaseRetrieve(setups, targetfft, gkp1, cache, fftresult); //fftresult is the starting point of the iteration
    return 0;
}
