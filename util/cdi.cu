#include "hip/hip_runtime.h"
#include <complex>
#include <tbb/tbb.h>
#include <fftw3-mpi.h>
# include <cassert>
# include <stdio.h>
# include <time.h>
# include <random>

#include <stdio.h>
#include "fftw.h"
#include <iostream>
#include <fstream>
#include <libconfig.h++>
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "hipfft/hipfft.h"

#include "common.h"
#include <ctime>

using std::cout; using std::endl;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

// This example reads the configuration file 'example.cfg' and displays
// some of its contents.
//#define Bits 16
__device__ __constant__ double cuda_beta_HIO;
__device__ __constant__ int cuda_row;
__device__ __constant__ int cuda_column;
__device__ __constant__ int cuda_rcolor;
__device__ __constant__ double cuda_scale;
using namespace cv;
double gaussian(double x, double y, double sigma){
  double r2 = pow(x,2) + pow(y,2);
  return exp(-r2/2/pow(sigma,2));
}

double gaussian_norm(double x, double y, double sigma){
  return 1./(2*pi*sigma*sigma)*gaussian(x,y,sigma);
}

enum mode {MOD2,MOD, REAL, IMAG, PHASE};
/******************************************************************************/

void maskOperation(Mat &input, Mat &output, Mat &kernel){
  filter2D(input, output, input.depth(), kernel);
}

template<typename T> using isInsideHandler = double(T::*)(double,double);

class support{
public:
  support(){};
  __device__ __host__ virtual bool isInside(int x, int y) = 0;
};
class ImageMask{
public:
  int nrow;
  int ncol;
  size_t sz;
  double *data;
  Mat *image;
  double threshold;
  ImageMask(){};
  void init_image(Mat* image_){
    nrow = image_->rows;
    ncol = image_->cols;
    image = image_;
    sz = image_->total()*sizeof(double);
    hipMalloc((void**)&data,sz);
  }
  void cpyToGM(){
    hipMemcpy(data, image->data, sz, hipMemcpyHostToDevice);
  }
  void cpyFromGM(){
    hipMemcpy(image->data, data, sz, hipMemcpyDeviceToHost);
  }
  __device__ __host__ bool isInside(int x, int y){
    if(data[x+y*nrow] < threshold) {
	    //printf("%d, %d = %f lower than threshold, dropping\n",x,y,image->ptr<double>(x)[y]);
	    return false;
    }
    return true;
  }
};
class rect : public support{
public:
  int startx;
  int starty;
  int endx;
  int endy;
  rect():support(){};
  __device__ __host__ bool isInside(int x, int y){
    if(x > startx && x <= endx && y > starty && y <= endy) return true;
    return false;
  }
};
class C_circle : public support{
public:
  int x0;
  int y0;
  double r;
  C_circle():support(){};
  __device__ __host__ bool isInside(int x, int y){
    double dr = sqrt(pow(x-x0,2)+pow(y-y0,2));
    if(dr < r) return true;
    return false;
  }
};
template<typename functor, typename format=fftw_complex>
void imageLoop(Mat* data, void* arg, bool isFrequency = 0){
  int row = data->rows;
  int column = data->cols;
  format *rowp;
  functor* func = static_cast<functor*>(arg);
  for(int x = 0; x < row ; x++){
    int targetx = x;
    if(isFrequency) targetx = x<row/2?x+row/2:(x-row/2);
    rowp = data->ptr<format>(x);
    for(int y = 0; y<column; y++){
      int targety = y;
      if(isFrequency) targety = y<column/2?y+column/2:(y-column/2);
      (*func)(targetx, targety , rowp[y]);
    }
  }
}
Mat* gaussianKernel(int rows, int cols, double sigma){
  Mat* image = new Mat(rows, cols, CV_64FC1);
  auto f = [&](int x, int y, double &data){
    data = gaussian_norm(x-rows/2,y-cols/2,sigma);
  };
  imageLoop<decltype(f), double>(image,&f);
  return image;
}
template<typename functor, typename format1, typename format2>
void imageLoop(Mat* data, Mat* dataout, void* arg, bool isFrequency = 0){
  int row = data->rows;
  int column = data->cols;
  format1 *rowp;
  format2 *rowo;
  functor* func = static_cast<functor*>(arg);
  for(int x = 0; x < row ; x++){
    int targetx = x;
    if(isFrequency) targetx = x<row/2?x+row/2:(x-row/2);
    rowp = data->ptr<format1>(x);
    rowo = dataout->ptr<format2>(targetx);
    for(int y = 0; y<column; y++){
      int targety = y;
      if(isFrequency) targety = y<column/2?y+column/2:(y-column/2);
      (*func)(targetx, targety , rowp[y], rowo[targety]);
    }
  }
}
/******************************************************************************/
double getVal(mode m, fftw_complex &data){
  complex<double> &tmpc = *(complex<double>*)(data);
  switch(m){
    case MOD:
      return std::abs(tmpc);
      break;
    case MOD2:
      return pow(std::abs(tmpc),2);
      break;
    case IMAG:
      return tmpc.imag();
      break;
    case PHASE:
      if(std::abs(tmpc)==0) return 0;
      return (std::arg(tmpc)+pi)/2/pi;
      break;
    default:
      return tmpc.real();
  }
}
double getVal(mode m, double &data){
  return data;
}
template<typename T=fftw_complex>
Mat* convertFromComplexToInteger(Mat *fftwImage, Mat* opencvImage = 0, mode m = MOD, bool isFrequency = 0, double decay = 1, const char* label= "default",bool islog = 0){
  pixeltype* rowo;
  T* rowp;
  int row = fftwImage->rows;
  int column = fftwImage->cols;
  if(!opencvImage) opencvImage = new Mat(row,column,format_cv);
  int tot = 0;
  double max = 0;
  for(int x = 0; x < row ; x++){
    int targetx = x;
    if(isFrequency) targetx = x<row/2?x+row/2:(x-row/2);
    rowo = opencvImage->ptr<pixeltype>(targetx);
    rowp = fftwImage->ptr<T>(x);
    for(int y = 0; y<column; y++){
      double target = getVal(m, rowp[y]);
      if(max < target) max = target;
      if(target<0) target = -target;
      if(islog){
        if(target!=0)
          target = log2(target)*rcolor/log2(rcolor)+rcolor;
	if(target < 0) target = 0;
	
      }
      else target*=rcolor*decay;

      tot += (int)target;
      if(target>=rcolor) {
	      //printf("larger than maximum of %s png %f\n",label, target);
	      target=rcolor-1;
	      //target=0;
      }
      int targety = y;
      if(isFrequency) targety = y<column/2?y+column/2:(y-column/2);
      rowo[targety] = floor(target);
      //if(opencv_reverted) rowp[targety] = rcolor - 1 - rowp[targety];
      //rowp[targety] = rcolor - 1 - rowp[targety];
    }
  }
  printf("total intensity %s: %d, max: %f\n", label, tot, max);
  return opencvImage;
}
template <typename inputtype>
Mat read16bitImage(Mat imagein, int nbitsimg)
{
    int row = imagein.rows;
    int column = imagein.cols;
    //int threshold = 1;
    int factor = pow(2,16-nbitsimg);
    Mat image(row/mergeDepth, column/mergeDepth, CV_16UC(1), Scalar::all(0));
    inputtype* rowp;
    uint16_t* rowo;
    int tot = 0;
    int max = 0;
    for(int x = 0; x < row ; x++){
	rowp = imagein.ptr<inputtype>(x);
	rowo = image.ptr<uint16_t>(x/mergeDepth);
        for(int y = 0; y<column; y++){
	    //if(rowp[y]>0) rowo[y]=rowp[y]/256;//log2(rowp[y])*pow(2,11);
	    //int nm1 = rowp[y-1];
	    //int np1 = rowp[y+1];
	    //if(n!=0) {
    	    //  int score0 = 4; // noise filter
	    //  if(y==0 || nm1<=threshold) score0--;
	    //  if(y==column-1 || np1<=threshold) score0--;
	    //  if(x==0 || imagein.ptr<inputtype>(x-1)[y]<=threshold) score0--;
	    //  if(x==row-1 || imagein.ptr<inputtype>(x+1)[y]<=threshold) score0--;
	    //  if(score0 > 1 || rowo[y] > threshold){
            //    //rowo[y]=floor(log2(n)*pow(2,12));//log2(rowp[y])*pow(2,11);
            //    if(nbits > inputbits) rowo[y]=n<<(nbits-inputbits);//log2(rowp[y])*pow(2,11);
	    //    else rowo[y]=n>>(inputbits-nbits);//log2(rowp[y])*pow(2,11);
	    //  }
	    //}
	    rowo[y/mergeDepth] += rowp[y]*factor/mergeDepth/mergeDepth;
	    tot+= rowp[y];
	    if(max < rowp[y]) max = rowp[y];
	}
    }
    printf("\ntot=%d,max=%d\n",tot,max);
    imwrite("input.png",image);
    return image;
}

Mat readImage(char* name, bool isFrequency = 0){
  Mat imagein = imread( name, IMREAD_UNCHANGED  );
  if(nbits == 8) {
   if(imagein.channels()==3){
     Mat image(imagein.rows, imagein.cols, format_cv);
     cv::cvtColor(imagein, image, cv::COLOR_BGR2GRAY);
     return image;
   }else{
     return imagein;
   }
  }
  if(imagein.depth() == CV_8U){
    printf("input image nbits: 8, channels=%d",imagein.channels());
    if(imagein.channels()>=3){
      Mat image(imagein.rows, imagein.cols, CV_8UC1);
      cv::cvtColor(imagein, image, cv::COLOR_BGR2GRAY);
      return read16bitImage<char>(image,8);
    }else
      return read16bitImage<char>(imagein,8);
  }else if(imagein.depth() == CV_16U){
    printf("input image nbits: 16");
    return read16bitImage<uint16_t>(imagein,16);
  }else{  //Image data is float
    printf("Image is not recognized as integer type, Image data is treated as floats\n");
    Mat *tmp = convertFromComplexToInteger<double>(&imagein,0,MOD,0,1,"input",1); //Here we save the logarithm of the input image
    imwrite("inputs.png", *tmp);
    delete tmp;
    Mat image(imagein.rows, imagein.cols, CV_64FC2);
    auto f = [&](int x, int y, double &data, fftw_complex &dataout){
      dataout[0] = max(0.,sqrt(data));
      dataout[1] = 0;
    };
    imageLoop<decltype(f),double,fftw_complex>(&imagein,&image,&f,1);
    return image;
  }

}

Mat* convertFromIntegerToComplex(Mat &image, Mat* cache = 0, bool isFrequency = 0, const char* label= "default"){
  int row = image.rows;
  int column = image.cols;
  if(!cache) cache = new Mat(row, column, CV_64FC2);
  double tot = 0;
  pixeltype* rowp;
  fftw_complex* rowo;
  int targetx, targety;
  for(int x = 0; x < row ; x++){
    if(isFrequency){
      targetx = x<row/2?x+row/2:(x-row/2);
    }else{
      targetx = x;
    }
    rowp = image.ptr<pixeltype>(x);
    rowo = cache->ptr<fftw_complex>(targetx);
    for(int y = 0; y<column; y++){
      if(isFrequency){
        targety = y<column/2?y+column/2:(y-column/2);
      }else{
	targety = y;
      }
      double intensity = ((double)rowp[y])/(rcolor-1);
      fftw_complex &datatmp = rowo[targety];
      if(opencv_reverted) intensity = 1-intensity;
      datatmp[0] = sqrt(intensity);
      datatmp[1] = 0;
      tot += sqrt(intensity);
    }
  }
  printf("total intensity %s: %f\n",label, tot);
  return cache;
}

Mat* convertFromIntegerToComplex(Mat &image,Mat &phase,Mat* cache = 0){
  int row = image.rows;
  int column = image.cols;
  if(!cache) cache = new Mat(row, column, CV_64FC2);
  int tot = 0;
  pixeltype *rowi, *rowp;
  fftw_complex *rowo;
  for(int x = 0; x < row ; x++){
    rowi = image.ptr<pixeltype>(x);
    rowp = phase.ptr<pixeltype>(x);
    rowo = phase.ptr<fftw_complex>(x);
    for(int y = 0; y<column; y++){
      tot += rowp[y];
      double phase = rowp[y];
      //phase*=2*pi/rcolor;
      //phase-=pi;
      phase = static_cast<double>(rand())/RAND_MAX*2*pi;
      rowo[y][0] = sqrt(((double)rowi[y])/rcolor)*cos(phase);
      rowo[y][1] = sqrt(((double)rowi[y])/rcolor)*sin(phase);
    }
  }
  printf("total intensity: %d\n", tot);
  return cache;
}

__global__ void applyNorm(hipfftDoubleComplex* data){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = x + y*cuda_row;
  data[index].x*=1./sqrtf(cuda_row*cuda_column);
  data[index].y*=1./sqrtf(cuda_row*cuda_column);
}

__global__ void applyMod(hipfftDoubleComplex* source, hipfftDoubleComplex* target, ImageMask *bs = 0){
  assert(source!=0);
  assert(target!=0);
  double tolerance = 0.5/cuda_rcolor*cuda_scale;
  double maximum = pow(mergeDepth,2)*cuda_scale;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = x + y*cuda_row;
  /*
  if(bs!=0){
    int tx = x;
    if(x >= cuda_row/2) tx -= cuda_row/2;
    else tx += cuda_row/2;
    int ty = y;
    if(y >= cuda_column/2) ty -= cuda_column/2;
    else ty += cuda_column/2;
    if(bs->isInside(tx,ty)) {
      return;
    }
  }
  */
  hipfftDoubleComplex targetdata = target[index];
  hipfftDoubleComplex sourcedata = source[index];
  double ratiox = 1;
  double ratioy = 1;
  double mod2 = targetdata.x*targetdata.x + targetdata.y*targetdata.y;
  double srcmod2 = sourcedata.x*sourcedata.x + sourcedata.y*sourcedata.y;
  //if(mod2>=maximum) {
  //  mod2 = max(maximum,srcmod2);
  //}
  double diff = mod2-srcmod2;
  if(diff>tolerance){
    ratioy=ratiox = sqrt((mod2-tolerance)/srcmod2);
  }else if(diff < -tolerance ){
    ratioy=ratiox = sqrt((mod2+tolerance)/srcmod2);
  }
  if(srcmod2 == 0){
    ratiox = sqrt(mod2);
    ratioy = 0;
  }
  source[index].x = ratiox*sourcedata.x;
  source[index].y = ratioy*sourcedata.y;
}

Mat* createWaveFront(Mat &intensity, Mat &phase, int rows, int columns, Mat* &itptr, Mat* wavefront = 0){
  if ( !intensity.data )
  {
      printf("No intensity data \n");
      exit(0);
  }
  if ( !phase.data )
  {
      printf("No phase data \n");
      exit(0);
  }
  if(intensity.rows!=phase.rows || intensity.cols!=phase.cols) {
    printf("intensity map and phase map having different dimensions");
    exit(0);
  }
  columns = intensity.cols;
  rows = intensity.rows;
  Mat *imageptr;
  itptr = &intensity;
  Mat &intensity_sc = *itptr;
  if(phase.channels()==3){
    imageptr = new Mat(rows, columns, format_cv);
    cv::cvtColor(phase, *imageptr, cv::COLOR_BGR2GRAY);
  }else{
    imageptr = &phase;
  }
  Mat &phase_sc = *imageptr;
  //wavefront = convertFromIntegerToComplex(intensity_sc, wavefront,0,"waveFront");
  wavefront = convertFromIntegerToComplex(intensity_sc, phase_sc, wavefront);
  delete imageptr;
  return wavefront;
  //imwrite("input.png",image);
}


Mat* extend( Mat &src , double ratio, double val = 0)
{
  Mat *dst = new Mat();
  int top, bottom, left, right;
  int borderType = BORDER_CONSTANT;
  if( src.empty()) {
      printf(" Error opening image\n");
      printf(" Program Arguments: [image_name -- default lena.jpg] \n");
      exit(0);
  }
  // Initialize arguments for the filter
  top = (int) ((ratio-1)/2*src.rows); bottom = top;
  left = (int) ((ratio-1)/2*src.cols); right = left;
  Scalar value(opencv_reverted?rcolor:0);
  copyMakeBorder( src, *dst, top, bottom, left, right, borderType, value );
  imwrite("ext.png",*dst);
  return dst;
}
void ApplyERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0];
    rhonp1[1] = rhoprime[1];
  }else{
    rhonp1[0] = rhonp1[1] = 0;
  }
}
void ApplyPOSERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0]*( rhoprime[0] > 0 );
    rhonp1[1] = 0;
  }else{
    rhonp1[0] = rhonp1[1] = 0;
  }
}
void ApplyLoosePOSERSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime, double threshold){
  if(rhoprime[0] < threshold){
    rhonp1[0] = rhoprime[0]*( rhoprime[0] > 0 );
  }else{
    rhonp1[0] = threshold;
  }
    rhonp1[1] = 0;
}
__device__ void ApplyHIOSupport(bool insideS, hipfftDoubleComplex &rhonp1, hipfftDoubleComplex &rhoprime, double beta){
  if(insideS){
    rhonp1.x = rhoprime.x;
    rhonp1.y = rhoprime.y;
  }else{
    rhonp1.x -= beta*rhoprime.x;
    rhonp1.y -= beta*rhoprime.y;
  }
}
void ApplyPOSHIOSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime, double beta){
  if(rhoprime[0] > 0 && (insideS/* || rhoprime[0]<30./rcolor*/)){
    rhonp1[0] = rhoprime[0];
    //rhonp1[1] = rhoprime[1];
    rhonp1[1] -= beta*rhoprime[1];
  }else{
    rhonp1[0] -= beta*rhoprime[0];
    rhonp1[1] -= beta*rhoprime[1];
  }
}
void ApplyLoosePOSHIOSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime, double beta, double threshold){
  if(rhoprime[0] > 0 && (rhoprime[0]<threshold)){
    rhonp1[0] = rhoprime[0];
    //rhonp1[1] = rhoprime[1];
    rhonp1[1] -= beta*rhoprime[1];
  }else{
    rhonp1[0] -= beta*(rhoprime[0]);
    rhonp1[1] -= beta*rhoprime[1];
  }
}
void ApplySFSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhoprime){
  if(insideS){
    rhonp1[0] = rhoprime[0];
    rhonp1[1] = rhoprime[1];
  }else{
    rhonp1[0] = -0.9*rhoprime[0];
    rhonp1[1] = -0.9*rhoprime[1];
  }
}
void ApplyDMSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhop, fftw_complex &pmsrho, double gammas, double gammam, double beta){

  complex<double> &rho = *(complex<double>*)rhonp1;
  complex<double> &rhoprime = *(complex<double>*)rhop;
  complex<double> &pmpsrho = *(complex<double>*)pmsrho;
  if(1||insideS){
    rho = 2.*pmpsrho-rhoprime;//(1-beta*gammam)*rhoprime+beta*(1+gammam+gammas)*rhoprime-beta*(1+gammas)*pmpsrho;
  }else{
    rho += 2.*pmpsrho-rhoprime;//beta*gammas*rhoprime-beta*(1+gammas)*pmpsrho;
  }
}
void ApplyPOSDMSupport(bool insideS, fftw_complex &rhonp1, fftw_complex &rhop, fftw_complex &pmsrho, double gammas, double gammam, double beta){

  complex<double> rho(rhonp1[0],rhonp1[1]);
  complex<double> rhoprime(rhop[0],rhop[1]);
  complex<double> pmpsrho(pmsrho[0],pmsrho[1]);
  if(insideS){
    rho = (1-beta*gammam)*rhoprime+beta*(1+gammam+gammas)*rhoprime-beta*(1+gammas)*pmpsrho;
  }else{
    rho += beta*gammas*rhoprime-beta*(1+gammas)*pmpsrho;
  }
  rhonp1[0] = rho.real();
  if(rhonp1[0]<0) rhonp1[0] = 0;
  rhonp1[1] = 0;
}
struct experimentConfig{
 bool useDM;
 bool useBS;
 bool useShrinkMap = 1;
 ImageMask* spt;
 ImageMask* beamStop;
 bool restart;
 double lambda = 0.6;
 double d = 16e3;
 double pixelsize = 6.5;
 double beamspotsize = 50;
};

__global__ void applySupport(hipfftDoubleComplex *gkp1, hipfftDoubleComplex *gkprime, double* objMod, ImageMask *spt){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = x + y*cuda_row;

  //epsilonF+=hypot(gkp1data[0]-gkprimedata[0],gkp1data[1]-gkprimedata[1]);
  //fftw_complex tmp = {gkp1data[0],gkp1data[1]};
  bool inside = spt->isInside(x,y);
  hipfftDoubleComplex &gkp1data = gkp1[index];
  hipfftDoubleComplex &gkprimedata = gkprime[index];
  //if(iter >= niters - 20 ) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20 || iter % 200 == 0) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20 || iter<20) ApplyERSupport(inside,gkp1data,gkprimedata);
  //if(iter >= niters - 20) ApplyERSupport(inside,gkp1data,gkprimedata);
  //ApplyERSupport(inside,gkp1data,gkprimedata);
  //else ApplyHIOSupport(inside,gkp1data,gkprimedata,beta_HIO);
  //else ApplyPOSHIOSupport(inside,gkp1data,gkprimedata,beta_HIO);
  //printf("%d, (%f,%f), (%f,%f), %f\n",inside, gkprimedata.x,gkprimedata.y,gkp1data.x,gkp1data.y,cuda_beta_HIO);
  ApplyHIOSupport(inside,gkp1data,gkprimedata,cuda_beta_HIO);
  objMod[index] = hipCabs(gkp1data);
  //double thres = gaussian(x-row/2,y-column/2,40);
  //ApplyLoosePOSHIOSupport(inside,gkp1data,gkprimedata,beta_HIO,thres);
  //ApplyLoosePOSERSupport(inside,gkp1data,gkprimedata,thres);
  //else {
  //ApplyDMSupport(inside,gkp1data, gkprimedata, pmpsg[index], gammas, gammam, beta);
  //}
  //ApplyERSupport(inside,pmpsg[index],gkp1data);
  //ApplyHIOSupport(inside,gkp1data,gkprimedata,beta);
  //else ApplySFSupport(inside,gkp1data,gkprimedata);
  //epsilonS+=hypot(tmp[0]-gkp1data[0],tmp[1]-gkp1data[1]);
}
void phaseRetrieve( experimentConfig &setups, Mat* targetfft, Mat* gkp1 = 0, Mat *cache = 0, Mat* fftresult = 0 ){
    Mat* pmpsg = 0;
    bool useShrinkMap = setups.useShrinkMap;
    int row = targetfft->rows;
    int column = targetfft->cols;
    bool useDM = setups.useDM;
    bool useBS = setups.useBS;
    ImageMask &re = *setups.spt;
    auto &cir = *(setups.beamStop);
    if(useDM) {
      pmpsg = new Mat();
      fftresult->copyTo(*pmpsg);
    }
    if(gkp1==0) gkp1 = new Mat(row,column,CV_64FC2);
    assert(targetfft!=0);
    double beta = -1;
    double beta_HIO = 0.9;
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_beta_HIO),&beta_HIO,sizeof(beta_HIO));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_row),&row,sizeof(row));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_column),&column,sizeof(column));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_rcolor),&rcolor,sizeof(rcolor));
    hipMemcpyToSymbol(HIP_SYMBOL(cuda_scale),&scale,sizeof(scale));
    double gammas = -1./beta;
    double gammam = 1./beta;
    double epsilonS, epsilonF;
    std::ofstream fepF,fepS;
    fepF.open("epsilonF.txt",ios::out |(setups.restart? ios::app:std::ios_base::openmode(0)));
    fepS.open("epsilonS.txt",ios::out |(setups.restart? ios::app:std::ios_base::openmode(0)));
    int niters = 5000;
    int tot = row*column;
    bool saveIter=1;
    Mat objMod(row,column,CV_64FC1);
    Mat* maskKernel;
    double gaussianSigma = 3;

    hipfftHandle *plan;
    plan = new hipfftHandle();
    hipfftPlan2d ( plan, row, column, HIPFFT_Z2Z);

    size_t sz = row*column*sizeof(hipfftDoubleComplex);
    hipfftDoubleComplex *cuda_fftresult, *cuda_targetfft, *cuda_gkprime, *cuda_gkp1, *cuda_pmpsg;
    double *cuda_objMod;
    ImageMask *cuda_spt;
    hipMalloc((void**)&cuda_fftresult, sz);
    hipMalloc((void**)&cuda_targetfft, sz);
    hipMalloc((void**)&cuda_gkprime, sz);
    hipMalloc((void**)&cuda_gkp1, sz);
    hipMalloc((void**)&cuda_objMod, sz/2);
    hipMalloc((void**)&cuda_spt, sizeof(ImageMask));
    hipMemcpy(cuda_spt, &re, sizeof(ImageMask), hipMemcpyHostToDevice);
    hipMemcpy(cuda_targetfft, targetfft->data, sz, hipMemcpyHostToDevice);
    hipMemcpy(cuda_fftresult, fftresult->data, sz, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16,16);
    dim3 numBlocks(row/threadsPerBlock.x, column/threadsPerBlock.y);

    hipfftExecZ2Z( *plan, cuda_targetfft, cuda_gkp1, HIPFFT_BACKWARD);
    applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_gkp1);
    hipDeviceSynchronize();
    std::chrono::time_point<std::chrono::high_resolution_clock> now = std::chrono::high_resolution_clock::now();
    std::chrono::duration<int64_t, std::nano> time_applyMod(0);
    std::chrono::duration<int64_t, std::nano> time_FFT(0);
    std::chrono::duration<int64_t, std::nano> time_support(0);
    std::chrono::duration<int64_t, std::nano> time_norm(0);
    for(int iter = 0; iter < niters; iter++){
      //start iteration
      if(iter%100==0 && saveIter) {
        hipMemcpy(gkp1->data, cuda_gkp1, sz, hipMemcpyDeviceToHost);
        printf("Iteration Number : %d\n", iter);
        convertFromComplexToInteger( gkp1,cache, MOD2,0);
        std::string iterstr = to_string(iter);
        imwrite("recon_intensity"+iterstr+".png",*cache);
        convertFromComplexToInteger( gkp1,cache, PHASE,0);
        imwrite("recon_phase"+iterstr+".png",*cache);
      }
      now = std::chrono::high_resolution_clock::now();
      if(useBS) applyMod<<<numBlocks,threadsPerBlock>>>(cuda_fftresult,cuda_targetfft,&cir);  //apply mod to fftresult, Pm
      else applyMod<<<numBlocks,threadsPerBlock>>>(cuda_fftresult,cuda_targetfft);  //apply mod to fftresult, Pm
      if(useDM) {
        if(useBS) applyMod<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg,cuda_targetfft,&cir);  
        else applyMod<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg,cuda_targetfft);
      }
      //hipDeviceSynchronize();
      time_applyMod+=std::chrono::high_resolution_clock::now()-now;
      
      epsilonS = epsilonF = 0;
      now = std::chrono::high_resolution_clock::now();
      hipfftExecZ2Z( *plan, cuda_fftresult, cuda_gkprime, HIPFFT_BACKWARD);
      time_FFT+=std::chrono::high_resolution_clock::now()-now;
      now = std::chrono::high_resolution_clock::now();
      applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_gkprime);
     // hipDeviceSynchronize();
      time_norm+=std::chrono::high_resolution_clock::now()-now;
      if(useDM){
        now = std::chrono::high_resolution_clock::now();
        hipfftExecZ2Z( *plan, cuda_pmpsg, cuda_pmpsg, HIPFFT_BACKWARD);
        time_FFT+=std::chrono::high_resolution_clock::now()-now;
        now = std::chrono::high_resolution_clock::now();
        applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg);
        hipDeviceSynchronize();
        time_norm+=std::chrono::high_resolution_clock::now()-now;
      }
      bool updateMask = iter%20==0 && useShrinkMap && iter!=0;
      if(updateMask){
        int size = floor(gaussianSigma*6); // r=3 sigma to ensure the contribution is negligible (0.01 of the maximum)
        size = size/2*2+1; //ensure odd
        maskKernel = gaussianKernel(size,size,gaussianSigma);
      }
      now = std::chrono::high_resolution_clock::now();
      applySupport<<<numBlocks,threadsPerBlock>>>(cuda_gkp1, cuda_gkprime, cuda_objMod,cuda_spt);
      time_support+=std::chrono::high_resolution_clock::now()-now;
      /*
        hipMemcpy(gkp1->data, cuda_gkp1, sz, hipMemcpyDeviceToHost);
        convertFromComplexToInteger( gkp1,cache, MOD2,0);
        imwrite("debug.png",*cache);
	*/
      //hipDeviceSynchronize();
      if(updateMask){
        hipMemcpy(objMod.data, cuda_objMod, sz/2, hipMemcpyDeviceToHost);
        filter2D(objMod, *re.image,objMod.depth(),*maskKernel);
	((ImageMask*)&re)->cpyToGM();
	if(gaussianSigma>1.5) gaussianSigma*=0.99;
	delete maskKernel;
      }
      if(updateMask&&iter%100==0&&saveIter){
	convertFromComplexToInteger<double>(re.image, cache,MOD,0);
        std::string iterstr = to_string(iter);
	imwrite("mask"+iterstr+".png",*cache);
      }
      if(iter!=0){
        fepF<<sqrt(epsilonF/tot)<<endl;
        fepS<<sqrt(epsilonS/tot)<<endl;
      }else {
        hipMemcpy(gkp1->data, gkp1, sz, hipMemcpyDeviceToHost);
        convertFromComplexToInteger( gkp1,cache, MOD2,0);
        imwrite("recon_support.png",*cache);
        convertFromComplexToInteger( gkp1,cache, PHASE,0);
        imwrite("recon_phase_support.png",*cache);
      }

      //if(sqrt(epsilonS/row/column)<0.05) break;
      now = std::chrono::high_resolution_clock::now();
      hipfftExecZ2Z( *plan, cuda_gkp1, cuda_fftresult, HIPFFT_FORWARD);
      time_FFT+=std::chrono::high_resolution_clock::now()-now;
      now = std::chrono::high_resolution_clock::now();
      applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_fftresult);
      //hipDeviceSynchronize();
      time_norm+=std::chrono::high_resolution_clock::now()-now;
      if(useDM){ // FFT to get f field;
        hipfftExecZ2Z( *plan, cuda_pmpsg, cuda_pmpsg, HIPFFT_FORWARD);
        applyNorm<<<numBlocks,threadsPerBlock>>>(cuda_pmpsg);
        hipDeviceSynchronize();
      }
      if(iter%100==0) {
	      long tot = time_FFT.count()+time_norm.count()+time_support.count()+time_applyMod.count();
	      printf("iter: %d, timing:\n  FFT:%ld, %f\n  NORM:%ld, %f\n  Support:%ld, %f\n  applyMod:%ld, %f\n",iter, 
			      time_FFT.count(),     ((double)time_FFT.count())/tot*100,
			      time_norm.count(),    ((double)time_norm.count())/tot*100,
			      time_support.count(), ((double)time_support.count())/tot*100,
			      time_applyMod.count(),((double)time_applyMod.count())/tot*100
              );
      }
      //end iteration
    }
    fepF.close();
    fepS.close();
    hipMemcpy(fftresult->data, cuda_fftresult, sz, hipMemcpyDeviceToHost);
    hipMemcpy(targetfft->data, cuda_targetfft, sz, hipMemcpyDeviceToHost);
    hipMemcpy(gkp1->data, gkp1, sz, hipMemcpyDeviceToHost);

    convertFromComplexToInteger( gkp1,cache, MOD2,0);
    imwrite("recon_intensity.png",*cache);
    convertFromComplexToInteger(gkp1, cache, PHASE,0);
    imwrite("recon_phase.png",*cache);
    if(useDM)  convertFromComplexToInteger( pmpsg, cache, MOD2,1);
    if(useDM)  imwrite("recon_pmpsg.png",*cache);
    convertFromComplexToInteger( fftresult, cache, MOD2,1);
    imwrite("recon_pattern.png",*cache);
}
/*
void autoCorrelationConstrain(Mat* pattern, support spt){
  Mat* autocorrelation = 0;
  for(int i = 0; i < 1000; i++){
    autocorrelation = fftw(pattern, autocorrelation, 0);
    for(int x = 0; x <pattern->row ; x++){
      patter->ptr
      spt.isInside(x,y);
    }
}
*/

int main(int argc, char** argv )
{

    if(argc < 2){
      printf("please feed the object intensity and phase image\n");
    }
    bool runSim;
    bool simCCDbit = 0;
    printf("command:");
    for(int i = 0; i < argc ; i++){
	    printf("%s ",argv[i]);
    }
    printf("\n");
    if(argv[1] == std::string("sim")){
      runSim = 1;
    }else{
      runSim = 0;
    }
    auto seed = (unsigned)time(NULL);
    bool isFresnel = 0;
    bool doIteration = 1;
    bool useGaussionLumination = 0;
    bool useGaussionHERALDO = 0;
    bool useRectHERALDO = 0;

    //int seed = 1657180926;// 1657180330
    //int seed = 1657182238; // oversampling = 3, modulation range = pi, reversed image
    //1657182741 // oversampling = 3, modulation range = 1.1pi, reversed image
    //1657182948 // oversampling = 3, modulation range = 1.3pi, reversed image
    //1657184141 // oversampling = 3, modulation range = 2pi, upright image, random phase
    srand(seed);
    printf("seed:%d\n",seed);
    double oversampling = 2;
    Mat* gkp1 = 0;
    Mat* targetfft = 0;
    Mat* fftresult = 0;
    bool restart = 0;
    if(argc > 4){
      restart = 1;
      
    }
    int row, column;
    Mat intensity = readImage( argv[2] , !runSim);
    //maskOperation(intensity,intensity);
    Mat ele = getStructuringElement(MORPH_RECT,Size(3,3),Point(1,1));
    //erode( intensity, intensity, ele);
    //dilate( intensity, intensity, ele);
    row = intensity.rows;
    column = intensity.cols;
    pixeltype *rowp;
    if(useRectHERALDO){
      for(int i = 0; i < row ; i++){
        rowp = intensity.ptr<pixeltype>(i);
        for(int j = 0; j < column ; j++){
          if(i > row/2 || j > column/2) rowp[j] = rcolor-1;
	}
      }
    }


    if(runSim){
	    row*=oversampling;
	    column*=oversampling;
    }
    C_circle cir,cir2,cir3,cir4;
    cir.x0=row/2;
    cir.y0=column/2;
    cir.r=10;
    //cir2.x0 = column*2/3-50;
    //cir2.y0 = row*2/3+110;
    //cir2.r = 50;
    cir2.x0 = column/2;
    cir2.y0 = row/2;
    cir2.r = 40;
    cir3.x0 = row/2;
    cir3.y0 = column/2;
    //cir3.r = 300/mergeDepth;
    cir3.r = 40;
    cir4.x0 = cir2.x0;
    cir4.y0 = cir2.y0;
    cir4.r = cir3.r;
    rect re;
    re.startx = (oversampling-1)/2*row/oversampling;
    re.starty = (oversampling-1)/2*column/oversampling;
    //re.startx = 1./4*row;
    //re.starty = 1./4*column;
    re.endx = row-re.startx;
    re.endy = column-re.starty;
    

    experimentConfig setups;
    setups.useShrinkMap = 1;
    ImageMask shrinkingMask;
    shrinkingMask.threshold = 0.1;
    setups.useDM = 0;
    setups.useBS = 0;

    setups.spt = &shrinkingMask;
    //setups.spt = &re;
    //setups.spt = &cir3;
    
    setups.beamStop = 0;//&cir;
    setups.restart = restart;
    //setups.d = oversampling*setups.pixelsize*setups.beamspotsize/setups.lambda; //distance to guarentee oversampling
    setups.pixelsize = 7;//setups.d/oversampling/setups.beamspotsize*setups.lambda;
    printf("recommanded imaging distance = %f\n", setups.d);
    printf("recommanded pixel size = %f\n", setups.pixelsize);

    bool isFarField = 0;
    double reversefresnelNumber = setups.d*setups.lambda/pi/pow(setups.beamspotsize,2);
    printf("Fresnel Number = %f\n",1./reversefresnelNumber);
    if(reversefresnelNumber > 100) isFarField = 1;
    //these are for simulation
    Mat* cache = 0;
    Mat* cache1;
    cache1 = &intensity;
    if(runSim){
      if(argc==4){
        Mat phase = readImage( argv[3]);
        if(oversampling>1) 
          gkp1 = createWaveFront(*extend(*cache1,oversampling), *extend(phase,oversampling), row, column,cache,gkp1);
        //if(oversampling>1) gkp1 = createWaveFront(extend(intensity,oversampling), extend(phase,oversampling), row, column,cache,gkp1);
        else gkp1 = createWaveFront(intensity,phase, row, column,cache,gkp1);
      }else{
        if(oversampling>1) cache = extend(*cache1,oversampling);
	else cache = cache1;
	gkp1 = convertFromIntegerToComplex(*cache, gkp1,0,"waveFront");
      }
      if(!isFarField && isFresnel){
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
	  double phase = pi*setups.lambda*setups.d/pow(setups.pixelsize,2)*(pow((x-0.5*row)/row,2)+pow((y-0.5*column)/column,2))/10;
	  *tmp *= exp(complex<double>(0,phase));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      if(useGaussionLumination){
        //setups.spt = &re;
        //if(!setups.useShrinkMap) setups.spt = &cir3;
        //diffraction image, either from simulation or from experiments.
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
          bool inside = cir3.isInside(x,y);
	  if(!inside) *tmp = 0.;
	  *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir3.r);
	  //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      if(useGaussionHERALDO){
        auto f = [&](int x, int y, fftw_complex &data){
          auto tmp = (complex<double>*)&data;
	  if(cir2.isInside(x,y)) 
		  *tmp *= gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
	  else *tmp = gaussian(x-cir2.x0,y-cir2.y0,cir2.r*4);
	  if(x < row*1/3 && y < row*1/3) *tmp = 0;
	  //if(cir2.isInside(x,y))printf("%f, ",gaussian(x-cir2.x0,y-cir2.y0,cir2.r/2));
	};
        imageLoop<decltype(f)>(gkp1,&f,0);
      }
      convertFromComplexToInteger(gkp1, cache, MOD2,0,1,"Object MOD2");
      imwrite("init_object.png",*cache);
      convertFromComplexToInteger(gkp1, cache, PHASE,0,1,"Object Phase");
      imwrite("init_object_phase.png",*cache);
      targetfft = fftw(gkp1,targetfft,1); 
    }else{
      //if(mergeDepth == 1) cache = cache1;
      //else 
      cache = new Mat(row, column, format_cv);
      if(cache1->depth() == CV_64F) 
        targetfft = cache1;
      else
        targetfft = convertFromIntegerToComplex(*cache1,targetfft,1); 
    }
    if(restart){
      intensity = readImage(argv[3]);
      Mat phase = readImage(argv[4]);
      gkp1 = createWaveFront(intensity, phase, row, column,cache,gkp1);
      fftresult = fftw(gkp1,fftresult,1); //If not restart, this line just allocate space, the values are not used.
    }
    //cir2.x0=row/2;
    //cir2.y0=column/2;
    double decay = scale;
    if(runSim) decay=1;
    std::default_random_engine generator;
    std::poisson_distribution<int> distribution(1000);
    Mat *autocorrelation = new Mat(row,column,CV_64FC2,Scalar::all(0.));
    shrinkingMask.init_image(new Mat(row,column,CV_64FC1));
    for(int i = 0; i<row*column; i++){ //remove the phase information
     // double randphase = arg(tmp);//static_cast<double>(rand())/RAND_MAX*2*pi;
      int tx = i/row;
      if(tx >= row/2) tx -= row/2;
      if(i/row < row/2) tx += row/2;
      int ty = i%row;
      if(ty >= column/2) ty -= column/2;
      if(i%row < column/2) ty += column/2;
      complex<double> &data = *(complex<double>*)((fftw_complex*)targetfft->data)[i];
      fftw_complex &datacor = ((fftw_complex*)autocorrelation->data)[i];
      double mod = abs(data)*sqrt(decay);
      if(runSim&&simCCDbit) {
        int range= pow(2,16);
        mod = sqrt(((double)floor(pow(mod,2)*range))/(range)); //assuming we use 16bit CCD
        //mod = sqrt(pow(mod,2)+double(distribution(generator))/range); //Poisson noise
      }
      if(1){
      if(setups.useBS && cir.isInside(tx,ty)) {
        data = 0.;
      }
      else{
        //complex<double> tmp(targetfft[i][0],targetfft[i][1]);
        double randphase = static_cast<double>(rand())/RAND_MAX*2*pi;
        data = mod*exp(complex<double>(0,randphase));
      }
      }
      //datacor[0] = pow(mod,2)*(tx-row/2)*(ty-column/2)/90; // ucore is the derivitaves of the diffraction pattern: append *(tx-row/2)*(ty-column/2)/20;
      datacor[0] = pow(mod,2); //ucore is the diffraction pattern
      datacor[1] = 0;
    }
    convertFromComplexToInteger( autocorrelation, cache, MOD,1,1,"HERALDO U core"); 
    imwrite("ucore.png",*cache);
    autocorrelation = fftw(autocorrelation, autocorrelation, 0);
    //autoCorrelationReconstruction(autocorrelation);

    auto f = [&](int x, int y, double &data, fftw_complex &dataout){
      data = hypot(dataout[1],dataout[0])>shrinkingMask.threshold;
    };
    imageLoop<decltype(f),double,fftw_complex>(shrinkingMask.image,autocorrelation,&f,1);
    convertFromComplexToInteger<double>(shrinkingMask.image, cache,MOD,0);
    imwrite("mask.png",*cache);
    shrinkingMask.cpyToGM();
    //auto f = [&](int x, int y, fftw_complex &data){
    //  auto tmp = (complex<double>*)&data;
    //  *tmp = 1.4+*tmp;
    //};
    //imageLoop<decltype(f)>(autocorrelation,&f,0);
    if(!restart){
      fftresult = new Mat();
      targetfft->copyTo(*fftresult);
    }
    convertFromComplexToInteger(targetfft, cache, PHASE,1);
    imwrite("init_phase.png",*cache);
    convertFromComplexToInteger(targetfft, cache, MOD2,1,1,"Pattern MOD2");
    imwrite("init_pattern.png",*cache);
    if(runSim) targetfft = convertFromIntegerToComplex(*cache, targetfft, 1 , "waveFront");
    convertFromComplexToInteger(autocorrelation, cache, MOD2,1,1,"Autocorrelation MOD2",1);
    imwrite("auto_correlation.png",*cache);
    //Phase retrieving starts from here. In the following, only targetfft is needed.
    if(doIteration) phaseRetrieve(setups, targetfft, gkp1, cache, fftresult); //fftresult is the starting point of the iteration
    return 0;
}
