#include "hip/hip_runtime.h"
#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <random>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <libconfig.h++>
#include "hipfft/hipfft.h"
#include "common.h"
#include <ctime>
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "tvFilter.h"
#include "cuPlotter.h"

#include <cub/device/device_reduce.cuh>
#define ALPHA 0.05
#define BETA 1
struct CustomMax
{
  __device__ __forceinline__
    complexFormat operator()(const complexFormat &a, const complexFormat &b) const {
      Real mod2a = a.x*a.x+a.y*a.y;
      Real mod2b = b.x*b.x+b.y*b.y;
      return (mod2a > mod2b) ? a : b;
    }
};

Real findMax(complexFormat* d_in, int num_items)
{
  complexFormat *d_out = NULL;
  hipMalloc((void**)&d_out, sizeof(complexFormat));

  void            *d_temp_storage = NULL;
  size_t          temp_storage_bytes = 0;
  CustomMax max_op;
  complexFormat tmp;
  tmp.x = tmp.y = 0;
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, tmp));
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, tmp));
  complexFormat output;
  hipMemcpy(&output, d_out, sizeof(complexFormat), hipMemcpyDeviceToHost);

  if (d_out) hipFree(d_out);
  if (d_temp_storage) hipFree(d_temp_storage);
  return output.x*output.x+output.y*output.y;
}

//#define Bits 16

Real gaussian(Real x, Real y, Real sigma){
  Real r2 = pow(x,2) + pow(y,2);
  return exp(-r2/2/pow(sigma,2));
}

Real gaussian_norm(Real x, Real y, Real sigma){
  return 1./(2*M_PI*sigma*sigma)*gaussian(x,y,sigma);
}

__global__ void multiplyProbe(complexFormat* object, complexFormat* probe, complexFormat* U, int shiftx, int shifty, int objrow, int objcol, complexFormat *window = 0){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  complexFormat tmp =  object[(x+shiftx)*objcol+y+shifty];
  if(window) window[index] = tmp;
  U[index] = hipCmulf(probe[index], tmp);
}

__global__ void updateObject(complexFormat* object, complexFormat* probe, complexFormat* U, int shiftx, int shifty, int objrow, int objcol, Real mod2maxProbe, Real mod2maxObj){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  int indexO = (x+shiftx)*objcol+y+shifty;
  complexFormat tmp = probe[index];
  complexFormat tmp2= object[indexO];
  Real mod2 = tmp.x*tmp.x+tmp.y*tmp.y;
  Real mod22 = tmp2.x*tmp2.x+tmp2.y*tmp2.y;
  tmp = hipCmulf(hipConjf(tmp), U[index]);
  tmp2 = hipCmulf(hipConjf(tmp2), U[index]);
  mod2 = (1-ALPHA)*mod2+ALPHA*mod2maxProbe;
  mod22 = (1-BETA)*mod22+BETA*mod2maxObj;
  tmp.x /= mod2;
  tmp.y /= mod2;
  object[indexO].x -= tmp.x;
  object[indexO].y -= tmp.y;
  tmp2.x /= mod22;
  tmp2.y /= mod22;
  probe[index].x -= tmp2.x;
  probe[index].y -= tmp2.y;
}

__global__ void random(complexFormat* object, hiprandStateMRG32k3a *state){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  hiprand_init(1,index,0,&state[index]);
  object[index].x = hiprand_uniform(&state[index]);
  object[index].y = hiprand_uniform(&state[index]);
}

class ptycho : public experimentConfig{
  public:
    int row_O = 1024;  //in ptychography this is different from row (the size of probe).
    int column_O = 1024;
    int sz = 0;
    int stepSize = 32;
    int dpupil = 100;
    int doPhaseModulationPupil = 0;
    int scanx = 0;
    int scany = 0;
    Real **patterns; //patterns[i*scany+j] points to the address on device to store pattern;
    complexFormat *esw;
    complexFormat *complexCache;

    ptycho(const char* configfile):experimentConfig(configfile){}
    void allocateMem(){
      printf("allocating memory\n");
      scanx = (row_O-row)/stepSize+1;
      scany = (column_O-column)/stepSize+1;
      printf("scanning %d x %d steps\n", scanx, scany);
      hipMalloc((void**)&objectWave, row_O*column_O*sizeof(Real)*2); //preallocation of support and beamstop
      hipMalloc((void**)&pupilpatternWave, sz*2);
      hipMalloc((void**)&esw, sz*2);
      hipMalloc((void**)&complexCache, sz*2);
      patterns = (Real**) malloc(scanx*scany*sizeof(Real*));
      for(int i = 0; i < scanx*scany ; i++) {
        hipMalloc((void**)&(patterns[i]),sz);
      }
      printf("initializing cuda image\n");
      init_cuda_image(row,column,rcolor,1./exposure);
    }
    void readPupilAndObject(){
      Real* object_intensity = readImage(common.Intensity.c_str(), row_O, column_O);
      Real* object_phase = readImage(common.Phase.c_str(), row_O, column_O);
      int objsz = row_O*column_O*sizeof(Real);
      Real* d_object_intensity;
      Real* d_object_phase;
      hipMalloc((void**)&d_object_intensity, objsz);
      hipMalloc((void**)&d_object_phase, objsz);
      hipMemcpy(d_object_intensity, object_intensity, objsz, hipMemcpyHostToDevice);
      hipMemcpy(d_object_phase, object_phase, objsz, hipMemcpyHostToDevice);
      free(object_intensity);
      free(object_phase);
      Real* pupil_intensity = readImage(pupil.Intensity.c_str(), row, column);
      sz = row*column*sizeof(Real);
      int row_tmp=row*oversampling;
      int column_tmp=column*oversampling;
      allocateMem();
      init_cuda_image(row_O,column_O,rcolor, 1./exposure);
      cudaF(createWaveFront)(d_object_intensity, d_object_phase, (complexFormat*)objectWave, 1);
      hipFree(d_object_intensity);
      hipFree(d_object_phase);
      verbose(2,
          plt.init(row_O,column_O);
          plt.plotComplex(objectWave, MOD2, 0, 1, "inputObject");
      )
      Real* d_intensity = patterns[0]; //use the memory allocated;
      hipMemcpy(d_intensity, pupil_intensity, sz, hipMemcpyHostToDevice);
      free(pupil_intensity);
      Real* d_phase = 0;
      if(doPhaseModulationPupil){
        d_phase = patterns[1];
        int tmp;
        Real* pupil_phase = readImage(pupil.Phase.c_str(), tmp,tmp);
        gpuErrchk(hipMemcpy(d_phase, pupil_phase, sz, hipMemcpyHostToDevice));
        free(pupil_phase);
      }
      hipMalloc((void**)&pupilobjectWave, row_tmp*column_tmp*sizeof(complexFormat));
      init_cuda_image(row_tmp,column_tmp,rcolor, 1./exposure);
      cudaF(createWaveFront)(d_intensity, d_phase, (complexFormat*)pupilobjectWave, oversampling);
      plt.init(row_tmp,column_tmp);
      plt.plotComplex(pupilobjectWave, MOD2, 0, 1, "pupilIntensity", 0);
      opticalPropagate((complexFormat*)pupilobjectWave, lambda, dpupil, beamspotsize*oversampling, row_tmp, column_tmp);
      init_cuda_image(row,column,rcolor, 1./exposure);
      cudaF(crop)((complexFormat*)pupilobjectWave, (complexFormat*)pupilpatternWave, row_tmp, column_tmp);
      plt.init(row,column);
      plt.plotComplex(pupilpatternWave, MOD2, 0, 1, "probeIntensity", 1);
      plt.plotComplex(pupilpatternWave, PHASE, 0, 1, "probePhase", 0);
      calculateParameters();
      multiplyFresnelPhase(pupilpatternWave, d);
    }
    void createPattern(){
      for(int i = 0; i < scanx; i++){
        for(int j = 0; j < scany; j++){
          cudaF(multiplyProbe)((complexFormat*)objectWave, (complexFormat*)pupilpatternWave, esw, i*stepSize, j*stepSize, row_O, column_O);
          verbose(3, plt.plotComplex(esw, MOD2, 0, 1, ("ptycho_esw"+to_string(i)+"_"+to_string(j)).c_str()));
          propagate(esw,esw,1);
          cudaF(getMod2)(patterns[i*scany+j], esw);
          verbose(3, plt.plotFloat(patterns[i*scany+j], MOD, 1, exposure, (common.Pattern+to_string(i)+"_"+to_string(j)).c_str()));
        }
      }
    }
    void initObject(){
      hiprandStateMRG32k3a *devstates;
      gpuErrchk(hipMalloc((void **)&devstates, column_O * row_O * sizeof(hiprandStateMRG32k3a)));
      init_cuda_image(row_O,column_O,rcolor, 1./exposure);
      cudaF(random)((complexFormat*)objectWave, devstates);
      init_cuda_image(row,column,rcolor, 1./exposure);
      //cudaF(random)((complexFormat*)pupilpatternWave, devstates);
      hipFree(devstates);
    }
    void iterate(){
      init_cuda_image(row,column,rcolor, 1./exposure);
      for(int iter = 0; iter < nIter; iter++){
        for(int i = 0; i < scanx; i++){
          for(int j = 0; j < scany; j++){
            int idx = i*scany+j;
            cudaF(multiplyProbe)((complexFormat*)objectWave, (complexFormat*)pupilpatternWave, esw,
              i*stepSize, j*stepSize, row_O, column_O, complexCache);
            Real objMax = findMax((complexFormat*)complexCache, row*column);
            propagate(esw,complexCache,1);
            cudaF(applyMod)(complexCache, patterns[idx]);
            propagate(complexCache,complexCache,0);
            cudaF(add)(esw, complexCache, -1);
            cudaF(updateObject)((complexFormat*)objectWave, (complexFormat*)pupilpatternWave, esw,
                i*stepSize, j*stepSize, row_O, column_O,//1,1);
                objMax,findMax((complexFormat*)pupilpatternWave, row*column));
          }
        }
      }
      plt.init(row, column);
      plt.plotComplex(pupilpatternWave, MOD2, 0, 1, "ptycho_probe_afterIter", 0);
      init_cuda_image(row_O,column_O,rcolor,1./exposure);
      plt.init(row_O, column_O);
      plt.plotComplex(objectWave, MOD2, 0, 1, "ptycho_afterIter", 0);
      plt.plotComplex(objectWave, PHASE, 0, 1, "ptycho_afterIterphase", 0);
    }
    void readPattern(){
      Real* pattern = readImage((common.Pattern+"0_0.png").c_str(), row, column);
      plt.init(row,column);
      sz = row*column*sizeof(Real);
      allocateMem();
      for(int i = 0; i < scanx; i++){
        for(int j = 0; j < scany; j++){
          int idx = i*scany+j;
          if(i!=0||j!=0) pattern = readImage((common.Pattern+to_string(i)+"_"+to_string(j)+".png").c_str(), row, column);
          hipMemcpy(patterns[idx], pattern, sz, hipMemcpyHostToDevice);
          free(pattern);
          cudaF(cudaConvertFO)(patterns[idx]);
          cudaF(applyNorm)(patterns[idx], 1./exposure);
          verbose(3, plt.plotFloat(patterns[i*scany+j], MOD, 1, exposure, ("input"+common.Pattern+to_string(i)+"_"+to_string(j)).c_str()));
        }
      }
      printf("Created pattern data\n");
      calculateParameters();
    }
    void calculateParameters(){
      resolution = lambda*dpupil/beamspotsize/oversampling;
      if(runSim) d = resolution*pixelsize*row/lambda;
      experimentConfig::calculateParameters();
    }
};
int main(int argc, char** argv )
{
  ptycho setups(argv[1]);
  hipFree(0); // to speed up the hipMalloc; https://forums.developer.nvidia.com/t/cudamalloc-slow/40238
  if(argc < 2){
    printf("please feed the object intensity and phase image\n");
  }
  setups.readPupilAndObject();
  printf("Imaging distance = %4.2fcm\n", setups.d*1e-4);
  printf("forward norm = %f\n", setups.forwardFactor);
  printf("backward norm = %f\n", setups.inverseFactor);
  printf("fresnel factor = %f\n", setups.fresnelFactor);
  printf("enhancement = %f\n", setups.enhancement);

  printf("pupil Imaging distance = %4.2fcm\n", setups.dpupil*1e-4);
  printf("pupil forward norm = %f\n", setups.forwardFactorpupil);
  printf("pupil backward norm = %f\n", setups.inverseFactorpupil);
  printf("pupil fresnel factor = %f\n", setups.fresnelFactorpupil);
  printf("pupil enhancement = %f\n", setups.enhancementpupil);
  setups.createPattern();
  setups.initObject();
  setups.iterate();

  return 0;
}

