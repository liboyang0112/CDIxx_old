#include "hip/hip_runtime.h"
#include <complex>
#include <cassert>
#include <stdio.h>
#include <time.h>
#include <random>
#include <chrono>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <libconfig.h++>
#include "hipfft/hipfft.h"
#include "common.h"
#include <ctime>
#include "cudaConfig.h"
#include "experimentConfig.h"
#include "tvFilter.h"
#include "cuPlotter.h"

#include <cub/device/device_reduce.cuh>
#define ALPHA 0.2
#define BETA 1
#define DELTA 1e-3
#define GAMMA 0.5
#define cudaIdx() \
int x = blockIdx.x * blockDim.x + threadIdx.x;\
int y = blockIdx.y * blockDim.y + threadIdx.y;\
if(x >= cuda_row || y >= cuda_column) return;\
int index = x*cuda_column + y;

struct CustomMax
{
  __device__ __forceinline__
    complexFormat operator()(const complexFormat &a, const complexFormat &b) const {
      Real mod2a = a.x*a.x+a.y*a.y;
      Real mod2b = b.x*b.x+b.y*b.y;
      return (mod2a > mod2b) ? a : b;
    }
};

Real findMax(complexFormat* d_in, int num_items)
{
  complexFormat *d_out = NULL;
  d_out = (complexFormat*)memMngr.borrowCache(sizeof(complexFormat));

  void            *d_temp_storage = NULL;
  size_t          temp_storage_bytes = 0;
  CustomMax max_op;
  complexFormat tmp;
  tmp.x = tmp.y = 0;
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, tmp));
  d_temp_storage = memMngr.borrowCache(temp_storage_bytes);

  // Run
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, max_op, tmp));
  complexFormat output;
  hipMemcpy(&output, d_out, sizeof(complexFormat), hipMemcpyDeviceToHost);

  if (d_out) memMngr.returnCache(d_out);
  if (d_temp_storage) memMngr.returnCache(d_temp_storage);
  return output.x*output.x + output.y*output.y;
}

struct CustomSumReal
{
  __device__ __forceinline__
    complexFormat operator()(const complexFormat &a, const complexFormat &b) const {
      return {a.x+b.x,0};
    }
};

Real findSumReal(complexFormat* d_in, int num_items)
{
  complexFormat *d_out = NULL;
  d_out = (complexFormat*)memMngr.borrowCache(sizeof(complexFormat));

  void            *d_temp_storage = NULL;
  size_t          temp_storage_bytes = 0;
  CustomSumReal sum_op;
  complexFormat tmp;
  tmp.x = 0;
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, sum_op, tmp));
  d_temp_storage = memMngr.borrowCache(temp_storage_bytes);

  // Run
  gpuErrchk(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, sum_op, tmp));
  complexFormat output;
  hipMemcpy(&output, d_out, sizeof(complexFormat), hipMemcpyDeviceToHost);

  if (d_out) memMngr.returnCache(d_out);
  if (d_temp_storage) memMngr.returnCache(d_temp_storage);
  return output.x;
}

//#define Bits 16

__device__ __host__ Real gaussian(Real x, Real y, Real sigma){
  Real r2 = pow(x,2) + pow(y,2);
  return exp(-r2/2/pow(sigma,2));
}

Real gaussian_norm(Real x, Real y, Real sigma){
  return 1./(2*M_PI*sigma*sigma)*gaussian(x,y,sigma);
}
__global__ void applySupport(Real* image, Real* support){
  cudaIdx();
  if(support[index] > cuda_threshold) image[index] = 0;
}


__global__ void multiplyProbe(complexFormat* object, complexFormat* probe, complexFormat* U, int shiftx, int shifty, int objrow, int objcol, complexFormat *window = 0){
  cudaIdx();
  complexFormat tmp;
  if(x+shiftx >= objrow || y+shifty >= objcol || x+shiftx < 0 || y+shifty < 0) tmp.x = tmp.y = 0;
  else tmp =  object[(x+shiftx)*objcol+y+shifty];
  if(window) window[index] = tmp;
  U[index] = hipCmulf(probe[index], tmp);
}

__global__ void getWindow(complexFormat* object, int shiftx, int shifty, int objrow, int objcol, complexFormat *window){
  cudaIdx();
  complexFormat tmp;
  if(x+shiftx >= objrow || y+shifty >= objcol || x+shiftx < 0 || y+shifty < 0) tmp.x = tmp.y = 0;
  else tmp =  object[(x+shiftx)*objcol+y+shifty];
  window[index] = tmp;
}

__global__ void updateWindow(complexFormat* object, int shiftx, int shifty, int objrow, int objcol, complexFormat *window){
  cudaIdx();
  if(x+shiftx >= objrow || y+shifty >= objcol || x+shiftx < 0 || y+shifty < 0) return;
  object[(x+shiftx)*objcol+y+shifty] = window[index];
}


__device__ void ePIE(complexFormat &target, complexFormat source, complexFormat &diff, Real maxi, Real param){
  Real denom = param/(maxi);
  source = hipCmulf(hipConjf(source),diff);
  target.x -= source.x*denom;
  target.y -= source.y*denom;
}

__device__ void rPIE(complexFormat &target, complexFormat source, complexFormat &diff, Real maxi, Real param){
  Real denom = source.x*source.x+source.y*source.y;
//  if(denom < 8e-4*maxi) return;
  denom = 1./((1-param)*denom+param*maxi);
  source = hipCmulf(hipConjf(source),diff);
  target.x -= source.x*denom;
  target.y -= source.y*denom;
}

__global__ void updateObject(complexFormat* object, complexFormat* probe, complexFormat* U, Real mod2maxProbe){
  cudaIdx()
  rPIE(object[index], probe[index], U[index], mod2maxProbe, ALPHA);
}

__global__ void updateObjectAndProbe(complexFormat* object, complexFormat* probe, complexFormat* U, Real mod2maxProbe, Real mod2maxObj){
  cudaIdx()
  complexFormat objectdat= object[index];
  complexFormat diff= U[index];
  rPIE(object[index], probe[index], diff, mod2maxProbe, ALPHA);
  rPIE(probe[index], objectdat, diff, mod2maxObj, BETA);
}

__global__ void random(complexFormat* object, hiprandStateMRG32k3a *state){
  cudaIdx()
  hiprand_init(1,index,0,&state[index]);
  object[index].x = hiprand_uniform(&state[index]);
  object[index].y = hiprand_uniform(&state[index]);
}

__global__ void pupilFunc(complexFormat* object){
  cudaIdx()
  int shiftx = x - cuda_row/2;
  int shifty = y - cuda_column/2;
  object[index].x = 3*gaussian(shiftx,shifty,cuda_row/8);
  object[index].y = 0;
}

__global__ void multiplyShift(complexFormat* object, Real shiftx, Real shifty){
  cudaIdx();
  Real phi = -2*M_PI*(shiftx*(x-cuda_row/2)/cuda_row+shifty*(y-cuda_column/2)/cuda_column);
  complexFormat tmp = {cos(phi),sin(phi)};
  object[index] = hipCmulf(object[index],tmp);
}

__global__ void multiplyx(complexFormat* object){
  cudaIdx();
  //object[index].x *= Real(x)/cuda_row;
  //object[index].y *= Real(x)/cuda_row;
  object[index].x *= Real(x)/cuda_row-0.5;
  object[index].y *= Real(x)/cuda_row-0.5;
}

__global__ void multiplyy(complexFormat* object){
  cudaIdx();
  //object[index].x *= Real(y)/cuda_row;
  //object[index].y *= Real(y)/cuda_row;
  object[index].x *= Real(y)/cuda_row-0.5;
  object[index].y *= Real(y)/cuda_row-0.5;
}

__global__ void calcPartial(complexFormat* object, complexFormat* Fn, Real* pattern, Real* beamstop){
  cudaIdx();
  if(beamstop[index] > 0.5){
    object[index].x = 0;
    return;
  }
  Real ret;
  auto fntmp = Fn[index];
  Real fnmod2 = fntmp.x*fntmp.x + fntmp.y*fntmp.y;
  ret = fntmp.x*object[index].y - fntmp.y*object[index].x;
  Real fact = pattern[index]+DELTA;
  if(fact<0) fact = 0;
  /*
  if(pattern[index]+DELTA<0) fact = 0;
  else fact = pow(pattern[index]+DELTA,GAMMA);
  ret*=(pow(fnmod2+DELTA,GAMMA)-fact);
  ret*=GAMMA*pow(fnmod2+DELTA,GAMMA-1);
  */
  ret*=1-sqrt(fact/(fnmod2+DELTA));
  //if(ret>1) printf("FIND larget ret %f at (%d, %d): (%f, %f), (%f, %f), %f, %f\n",ret, x, y, object[index].x, object[index].y, fntmp.x, fntmp.y, pattern[index], beamstop[index]);
  object[index].x = ret;
}

void shiftWave(complexFormat* wave, int npix, Real shiftx, Real shifty){
  myCufftExec( *plan, wave, wave, HIPFFT_FORWARD);
  cudaF(cudaConvertFO)(wave);
  cudaF(multiplyShift)(wave, shiftx, shifty);
  cudaF(cudaConvertFO)(wave);
  myCufftExec( *plan, wave, wave, HIPFFT_BACKWARD);
  cudaF(applyNorm)(wave, 1./npix);
}

class ptycho : public experimentConfig{
  public:
    int row_O = 512;  //in ptychography this is different from row (the size of probe).
    int column_O = 512;
    int sz = 0;
    int stepSize = 32;
    int doPhaseModulationPupil = 0;
    int scanx = 0;
    int scany = 0;
    Real *shiftx = 0;
    Real *shifty = 0;
    Real **patterns; //patterns[i*scany+j] points to the address on device to store pattern;
    complexFormat *esw;
    hiprandStateMRG32k3a *devstates = 0;

    ptycho(const char* configfile):experimentConfig(configfile){}
    void allocateMem(){
      if(devstates) return;
      devstates = (hiprandStateMRG32k3a*) memMngr.borrowCache(column_O * row_O * sizeof(hiprandStateMRG32k3a));
      printf("allocating memory\n");
      scanx = (row_O-row)/stepSize+1;
      scany = (column_O-column)/stepSize+1;
      printf("scanning %d x %d steps\n", scanx, scany);
      objectWave = (complexFormat*)memMngr.borrowCache(row_O*column_O*sizeof(Real)*2);
      pupilpatternWave = (complexFormat*)memMngr.borrowCache(sz*2);
      esw = (complexFormat*) memMngr.borrowCache(sz*2);
      patterns = (Real**) malloc(scanx*scany*sizeof(Real*));
      memset(patterns, 0, scanx*scany*sizeof(Real*)/sizeof(char));
      printf("initializing cuda image\n");
      init_cuda_image(row_O,column_O,rcolor,1./exposure);
      cudaF(initRand)(devstates);
      if(positionUncertainty > 1e-4){
        initPosition();
      }
    }
    void readPupilAndObject(){
      Real* object_intensity = readImage(common.Intensity.c_str(), row_O, column_O);
      Real* object_phase = readImage(common.Phase.c_str(), row_O, column_O);
      int objsz = row_O*column_O*sizeof(Real);
      Real* d_object_intensity;
      Real* d_object_phase;
      d_object_intensity = (Real*)memMngr.borrowCache(objsz);
      d_object_phase = (Real*)memMngr.borrowCache(objsz);
      hipMemcpy(d_object_intensity, object_intensity, objsz, hipMemcpyHostToDevice);
      hipMemcpy(d_object_phase, object_phase, objsz, hipMemcpyHostToDevice);
      ccmemMngr.returnCache(object_intensity);
      ccmemMngr.returnCache(object_phase);
      Real* pupil_intensity = readImage(pupil.Intensity.c_str(), row, column);
      sz = row*column*sizeof(Real);
      int row_tmp=row*oversampling;
      int column_tmp=column*oversampling;
      allocateMem();
      cudaF(createWaveFront)(d_object_intensity, d_object_phase, (complexFormat*)objectWave, 1);
      memMngr.returnCache(d_object_intensity);
      memMngr.returnCache(d_object_phase);
      verbose(2,
          plt.init(row_O,column_O);
          plt.plotComplex(objectWave, MOD2, 0, 1, "inputObject");
          plt.plotComplex(objectWave, PHASE, 0, 1, "inputPhase");
          //plt.plotPhase(objectWave, PHASERAD, 0, 1, "inputPhase");
      )
      Real* d_intensity = (Real*) memMngr.borrowCache(sz); //use the memory allocated;
      hipMemcpy(d_intensity, pupil_intensity, sz, hipMemcpyHostToDevice);
      ccmemMngr.returnCache(pupil_intensity);
      Real* d_phase = 0;
      if(doPhaseModulationPupil){
        d_phase = (Real*) memMngr.borrowCache(sz);
        int tmp;
        Real* pupil_phase = readImage(pupil.Phase.c_str(), tmp,tmp);
        gpuErrchk(hipMemcpy(d_phase, pupil_phase, sz, hipMemcpyHostToDevice));
        ccmemMngr.returnCache(pupil_phase);
      }
      pupilobjectWave = (complexFormat*)memMngr.borrowCache(row_tmp*column_tmp*sizeof(complexFormat));
      init_cuda_image(row_tmp,column_tmp,rcolor, 1./exposure);
      cudaF(createWaveFront)(d_intensity, d_phase, (complexFormat*)pupilobjectWave, oversampling);
      memMngr.returnCache(d_intensity);
      if(d_phase) memMngr.returnCache(d_phase);
      plt.init(row_tmp,column_tmp);
      plt.plotComplex(pupilobjectWave, MOD2, 0, 1, "pupilIntensity", 0);
      init_fft(row_tmp,column_tmp);
      opticalPropagate((complexFormat*)pupilobjectWave, lambda, dpupil, beamspotsize*oversampling, row_tmp, column_tmp);
      plt.plotComplex(pupilobjectWave, MOD2, 0, 1, "pupilPattern", 0);
      init_cuda_image(row,column,rcolor, 1./exposure);
      init_fft(row,column);
      cudaF(crop)((complexFormat*)pupilobjectWave, (complexFormat*)pupilpatternWave, row_tmp, column_tmp);
      plt.init(row,column);
      plt.plotComplex(pupilpatternWave, MOD2, 0, 1, "probeIntensity", 0);
      plt.plotComplex(pupilpatternWave, PHASE, 0, 1, "probePhase", 0);
      calculateParameters();
      multiplyFresnelPhase(pupilpatternWave, d);
    }
    void initPosition(){
      shiftx = (Real*)ccmemMngr.borrowCache(scanx*scany*sizeof(Real));
      shifty = (Real*)ccmemMngr.borrowCache(scanx*scany*sizeof(Real));
      memset(shiftx, 0, scanx*scany*sizeof(Real)/sizeof(char));
      memset(shifty, 0, scanx*scany*sizeof(Real)/sizeof(char));
      if(runSim && positionUncertainty>1e-4){
        unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::default_random_engine generator(seed);
        std::normal_distribution<double> distribution(0.0, 1.);
        for(int i = 0 ; i < scanx*scany; i++){
          shiftx[i]+= distribution(generator)*positionUncertainty;
          shifty[i]+= distribution(generator)*positionUncertainty;
          printf("shifts (%d, %d): (%f, %f)\n", i/scany, i%scany, shiftx[i],shifty[i]);
        }
      }
    }
    void resetPosition(){
      for(int i = 0 ; i < scanx*scany; i++){
        shiftx[i] = shifty[i] = 0;
      }
    }
    void createPattern(){
      int idx = 0;
      if(useBS) {
        createBeamStop();
        plt.plotFloat(beamstop, MOD, 1, 1,"beamstop", 0);
      }
      complexFormat* window = (complexFormat*)memMngr.borrowCache(sz*2);
      for(int i = 0; i < scanx; i++){
        for(int j = 0; j < scany; j++){
          int shiftxpix = shiftx[idx]-round(shiftx[idx]);
          int shiftypix = shiftx[idx]-round(shifty[idx]);
          cudaF(getWindow)((complexFormat*)objectWave, i*stepSize-round(shiftx[idx]), j*stepSize-round(shifty[idx]), row_O, column_O, window);
          if(fabs(shiftxpix)>1e-3||fabs(shiftypix)>1e-3){
            shiftWave((complexFormat*)window, row*column, shiftxpix, shiftypix);
          }
          cudaF(multiply)(esw, (complexFormat*)pupilpatternWave, window);
          verbose(5, plt.plotComplex(esw, MOD2, 0, 1, ("ptycho_esw"+to_string(i)+"_"+to_string(j)).c_str()));
          propagate(esw,esw,1);
          if(!patterns[idx]) patterns[idx] = (Real*)memMngr.borrowCache(sz);
          cudaF(getMod2)(patterns[idx], esw);
          if(useBS) cudaF(applySupport)(patterns[idx], beamstop);
          if(simCCDbit) cudaF(applyPoissonNoise_WO)(patterns[idx], noiseLevel, devstates, 1./exposure);
          verbose(3, plt.plotFloat(patterns[idx], MOD, 1, exposure, (common.Pattern+to_string(i)+"_"+to_string(j)).c_str()));
          verbose(4, plt.plotFloat(patterns[idx], MOD, 1, exposure, (common.Pattern+to_string(i)+"_"+to_string(j)+"log").c_str(),1));
          idx++;
        }
      }
      memMngr.returnCache(window);
    }
    void initObject(){
      init_cuda_image(row_O,column_O,rcolor, 1./exposure);
      cudaF(random)((complexFormat*)objectWave, devstates);
      init_cuda_image(row,column,rcolor, 1./exposure);
      cudaF(pupilFunc)((complexFormat*)pupilpatternWave);
    }
    void updatePosition(Real &shiftx, Real &shifty, complexFormat* obj, complexFormat* probe, Real* pattern, complexFormat* Fn){
      Real siz = memMngr.getSize(obj);
      complexFormat *cachex = (complexFormat*)memMngr.borrowCache(siz);
      complexFormat *cachey = (complexFormat*)memMngr.borrowCache(siz);
      propagate(obj, cachex, 1);
      cudaF(cudaConvertFO)(cachex);
      hipMemcpy(cachey, cachex, siz, hipMemcpyDeviceToDevice);
      cudaF(multiplyx)(cachex);
      cudaF(multiplyy)(cachey);
      cudaF(cudaConvertFO)(cachex);
      cudaF(cudaConvertFO)(cachey);
      propagate(cachex, cachex, 0);
      propagate(cachey, cachey, 0);
      cudaF(multiply)(cachex, probe);
      cudaF(multiply)(cachey, probe);
      propagate(cachex, cachex, 1);
      propagate(cachey, cachey, 1);
      cudaF(calcPartial)(cachex, Fn, pattern, beamstop);
      cudaF(calcPartial)(cachey, Fn, pattern, beamstop);
      Real partialx = 8*M_PI*findSumReal(cachex,row*column);
      Real partialy = 8*M_PI*findSumReal(cachey,row*column);
      memMngr.returnCache(cachex);
      memMngr.returnCache(cachey);
      shiftx -= partialx*1e-3;
      shifty -= partialy*1e-3;
      if(shiftx!=shiftx || shifty!=shifty) exit(0);
    }
    void iterate(){
      resetPosition();
      init_cuda_image(row,column,rcolor, 1./exposure);
      Real objMax;
      complexFormat *Fn = (complexFormat*)memMngr.borrowCache(sz*2);
      complexFormat *objCache = (complexFormat*)memMngr.borrowCache(sz*2);
      int update_probe_iter = 4;
      for(int iter = 0; iter < nIter; iter++){
        int idx = 0;
        for(int i = 0; i < scanx; i++){
          for(int j = 0; j < scany; j++){
            int shiftxpix = shiftx[idx]-round(shiftx[idx]);
            int shiftypix = shiftx[idx]-round(shifty[idx]);
            cudaF(getWindow)((complexFormat*)objectWave,
                i*stepSize-round(shiftx[idx]), j*stepSize-round(shifty[idx]), row_O, column_O, objCache);
            if(fabs(shiftxpix)>1e-3||fabs(shiftypix)>1e-3){
              shiftWave((complexFormat*)objCache, row*column, shiftxpix, shiftypix);
            }
            cudaF(multiply)(esw, (complexFormat*)pupilpatternWave, objCache);
            if(iter >= update_probe_iter) objMax = findMax((complexFormat*)objCache, row*column);
            propagate(esw,Fn,1);
            if(iter % 100 == 0 && iter >= 100) updatePosition(shiftx[idx], shifty[idx], objCache, (complexFormat*)pupilpatternWave, patterns[idx], Fn);
            //if(iter > 150 && i == 4 && j == 4) updatePosition(shiftx[idx], shifty[idx], objCache, (complexFormat*)pupilpatternWave, patterns[idx], Fn);
            Real probeMax = findMax((complexFormat*)pupilpatternWave, row*column);
            cudaF(applyMod)(Fn, patterns[idx],beamstop,1);
            propagate(Fn,Fn,0);
            cudaF(add)(esw, Fn, -1);
            if(iter < update_probe_iter) cudaF(updateObject)(objCache, (complexFormat*)pupilpatternWave, esw,//1,1);
                probeMax);
            else cudaF(updateObjectAndProbe)(objCache, (complexFormat*)pupilpatternWave, esw,//1,1);
                probeMax, objMax);
            if(fabs(shiftxpix)>1e-3||fabs(shiftypix)>1e-3){
              shiftWave(objCache, row*column, -shiftxpix, -shiftypix);
            }
            cudaF(updateWindow)((complexFormat*)objectWave,
                i*stepSize-round(shiftx[idx]), j*stepSize-round(shifty[idx]), row_O, column_O, objCache);
            idx++;
          }
        }
        if(iter == 200){
          init_cuda_image(row_O,column_O,rcolor,1./exposure);
          plt.init(row_O, column_O);
          plt.plotComplex(objectWave, MOD2, 0, 1, "ptycho_b4position", 0);
          plt.plotComplex(objectWave, PHASE, 0, 1, "ptycho_b4positionphase", 0);
          init_cuda_image(row,column,rcolor,1./exposure);
          plt.init(row, column);
        }
      }
      for(int i = 0 ; i < scanx*scany; i++){
        printf("recon shifts (%d, %d): (%f, %f)\n", i/scany, i%scany, shiftx[i],shifty[i]);
      }
      memMngr.returnCache(Fn);
      memMngr.returnCache(objCache);
      plt.init(row, column);
      plt.plotComplex(pupilpatternWave, MOD2, 0, 1, "ptycho_probe_afterIter", 0);
      init_cuda_image(row_O*4/7,column_O*4/7,rcolor,1./exposure);
      plt.init(row_O*4/7, column_O*4/7);
      complexFormat* output = (complexFormat*)memMngr.borrowCache(row_O*2/3*(column_O*4/7)*sizeof(complexFormat));
      cudaF(crop)((complexFormat*)objectWave, output, row_O, column_O);
      plt.plotComplex(output, MOD2, 0, 1.5/findMax(output, row_O*4/7*(column_O*4/7)), "ptycho_afterIter", 0);
      //plt.plotComplex(objectWave, PHASE, 0, 1, "ptycho_afterIterphase", 0);
      plt.plotPhase(output, PHASERAD, 0, 1, "ptycho_afterIterphase", 0);
    }
    void readPattern(){
      Real* pattern = readImage((common.Pattern+"0_0.png").c_str(), row, column);
      plt.init(row,column);
      init_fft(row,column);
      sz = row*column*sizeof(Real);
      allocateMem();
      init_cuda_image(row,column,rcolor, 1./exposure);
      createBeamStop();
      int idx = 0;
      for(int i = 0; i < scanx; i++){
        for(int j = 0; j < scany; j++){
          if(idx!=0) pattern = readImage((common.Pattern+to_string(i)+"_"+to_string(j)+".png").c_str(), row, column);
          if(!patterns[idx]) patterns[idx] = (Real*)memMngr.borrowCache(sz);
          hipMemcpy(patterns[idx], pattern, sz, hipMemcpyHostToDevice);
          ccmemMngr.returnCache(pattern);
          cudaF(cudaConvertFO)(patterns[idx]);
          cudaF(applyNorm)(patterns[idx], 1./exposure);
          verbose(3, plt.plotFloat(patterns[idx], MOD, 1, exposure, ("input"+common.Pattern+to_string(i)+"_"+to_string(j)).c_str()));
          idx++;
        }
      }
      printf("Created pattern data\n");
      calculateParameters();
    }
    void calculateParameters(){
      resolution = lambda*dpupil/beamspotsize/oversampling;
      if(runSim) d = resolution*pixelsize*row/lambda;
      experimentConfig::calculateParameters();
    }
};
int main(int argc, char** argv )
{
  ptycho setups(argv[1]);
  hipFree(0); // to speed up the hipMalloc; https://forums.developer.nvidia.com/t/cudamalloc-slow/40238
  if(argc < 2){
    printf("please feed the object intensity and phase image\n");
  }
  if(setups.runSim){
    setups.readPupilAndObject();
    setups.createPattern();
  }else{
    setups.readPattern();
  }
  printf("Imaging distance = %4.2fcm\n", setups.d*1e-4);
  printf("fresnel factor = %f\n", setups.fresnelFactor);
  printf("Resolution = %4.2fnm\n", setups.resolution*1e3);

  printf("pupil Imaging distance = %4.2fcm\n", setups.dpupil*1e-4);
  printf("pupil fresnel factor = %f\n", setups.fresnelFactorpupil);
  printf("pupil enhancement = %f\n", setups.enhancementpupil);
  setups.initObject();
  setups.iterate();

  return 0;
}

