#include "hip/hip_runtime.h"
#include "format.h"
#include "cudaConfig.h"
#include <hipfft/hipfftw.h>
#include <iostream>
using namespace cv;
using namespace std;

void fftw_init(){
  
}
void Check(hipError_t status)
{
        if (status != hipSuccess)
        {
                cout << "行号:" << __LINE__ << endl;
                cout << "错误:" << hipGetErrorString(status) << endl;
        }
}

static complexFormat *cudaData = 0;
static hipfftHandle *plan; 
static size_t sz;

Mat* fftw ( Mat* in, Mat *out, bool isforward, Real ratio)
{
  int row = in->rows;
  int column = in->cols;
  if(ratio==0) ratio = 1./sqrt(row*column);
  if(out == 0) out = new Mat(row,column,float_cv_format(2));

  if(cudaData==0) {
    sz = row*column*sizeof(complexFormat);
    Check(hipMalloc((void**)&cudaData, sz));
    plan = new hipfftHandle();
    hipfftPlan2d ( plan, row, column, FFTformat);
  }else{
    if(sz!=row*column*sizeof(complexFormat)){
      printf("reconfiguring CUFFT\n");
      sz = row*column*sizeof(complexFormat);
      hipFree(cudaData);
      Check(hipMalloc((void**)&cudaData, sz));
      hipfftPlan2d ( plan, row, column, FFTformat);
    }
  }
  Check(hipMemcpy(cudaData, in->data, sz, hipMemcpyHostToDevice));
    
  myCufftExec( *plan, cudaData,cudaData, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);

  Check(hipMemcpy(out->data, cudaData, sz, hipMemcpyDeviceToHost));

  for(int i = 0; i < out->total() ; i++){
    ((complexFormat*)out->data)[i].x*=ratio;
    ((complexFormat*)out->data)[i].y*=ratio;
  } //normalization
  //hipfftDestroy(*plan);
  return out;
}

