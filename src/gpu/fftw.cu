#include "hip/hip_runtime.h"
#include "fftw.h"
#include <hipfft/hipfftw.h>
#include <iostream>
using namespace cv;
using namespace std;

void fftw_init(){
  
}
void Check(hipError_t status)
{
        if (status != hipSuccess)
        {
                cout << "行号:" << __LINE__ << endl;
                cout << "错误:" << hipGetErrorString(status) << endl;
        }
}

static hipfftDoubleComplex *cudaData = 0;
static hipfftHandle *plan; 
static size_t sz;

Mat* fftw ( Mat* in, Mat *out = 0, bool isforward = 1)
{
  int row = in->rows;
  int column = in->cols;
  double ratio = 1./sqrt(row*column);
  if(out == 0) out = new Mat(row,column,CV_64FC2);

  if(cudaData==0) {
    sz = row*column*sizeof(hipfftDoubleComplex);
    Check(hipMalloc((void**)&cudaData, sz));
    plan = new hipfftHandle();
    hipfftPlan2d ( plan, row, column, HIPFFT_Z2Z);
  }else{
    if(sz!=row*column*sizeof(hipfftDoubleComplex)){
      sz = row*column*sizeof(hipfftDoubleComplex);
      hipFree(cudaData);
      Check(hipMalloc((void**)&cudaData, sz));
      hipfftPlan2d ( plan, row, column, HIPFFT_Z2Z);
    }
  }
  Check(hipMemcpy(cudaData, in->data, sz, hipMemcpyHostToDevice));
    
  hipfftExecZ2Z( *plan, cudaData,cudaData, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);

  Check(hipMemcpy(out->data, cudaData, sz, hipMemcpyDeviceToHost));

  for(int i = 0; i < out->total() ; i++){
    ((hipfftDoubleComplex*)out->data)[i].x*=ratio;
    ((hipfftDoubleComplex*)out->data)[i].y*=ratio;
  } //normalization
  //hipfftDestroy(*plan);
  return out;
}

