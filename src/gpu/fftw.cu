#include "hip/hip_runtime.h"
#include <hipfft/hipfftw.h>
#include "fftw.h"
#include <iostream>
using namespace cv;
using namespace std;

void fftw_init(){
  
}
void Check(hipError_t status)
{
        if (status != hipSuccess)
        {
                cout << "行号:" << __LINE__ << endl;
                cout << "错误:" << hipGetErrorString(status) << endl;
        }
}

static hipfftDoubleComplex *cudaData = 0;
static hipfftHandle *plan; 
static size_t sz;

Mat* fftw ( Mat* in, Mat *out = 0, bool isforward = 1)
{
  int row = in->rows;
  int column = in->cols;
  double ratio = 1./sqrt(row*column);
  if(out == 0) out = new Mat(row,column,CV_64FC2);

  if(cudaData==0) {
    sz = row*column*sizeof(hipfftDoubleComplex);
    Check(hipMalloc((void**)&cudaData, sz));
    plan = new hipfftHandle();
    hipfftPlan2d ( plan, row, column, HIPFFT_Z2Z);
  }else{
    if(sz!=row*column*sizeof(hipfftDoubleComplex)){
      printf("ERROR: currently cufft only supports single image size to avoid construct and destroy the plan, please check if you are trying to FFT images with different dimensions:\n %lu/(%d*%d*%lu)=%f\n",sz,row,column,sizeof(hipfftDoubleComplex),((double)sz)/row/column/sizeof(hipfftDoubleComplex));
      printf("FILE: %s, LINE: %d\n",__FILE__, __LINE__);
      exit(0);
    }
  }
  Check(hipMemcpy(cudaData, in->data, sz, hipMemcpyHostToDevice));
    
  hipfftExecZ2Z( *plan, cudaData,cudaData, isforward? HIPFFT_FORWARD: HIPFFT_BACKWARD);

  Check(hipMemcpy(out->data, cudaData, sz, hipMemcpyDeviceToHost));

  for(int i = 0; i < out->total() ; i++){
    ((hipfftDoubleComplex*)out->data)[i].x*=ratio;
    ((hipfftDoubleComplex*)out->data)[i].y*=ratio;
  } //normalization
  //hipfftDestroy(*plan);
  return out;
}

