#include "hip/hip_runtime.h"
#include "cuPlotter.h"
#include "cudaDefs.h"
#include <cassert>

void cuPlotter::initcuData(size_t sz){
  if(cuCache_data) memMngr.returnCache(cuCache_data);
  if(cuCache_float_data) memMngr.returnCache(cuCache_float_data);
  if(cuCache_complex_data) memMngr.returnCache(cuCache_complex_data);
  cuCache_data = (pixeltype*) memMngr.borrowCache(sz*sizeof(pixeltype));
  cuCache_float_data = (Real*) memMngr.borrowCache(sz*sizeof(Real));
  cuCache_complex_data = (complexFormat*) memMngr.borrowCache(sz*sizeof(complexFormat));
}

void cuPlotter::freeCuda(){
  if(cuCache_data) memMngr.returnCache(cuCache_data);
  if(cuCache_float_data) memMngr.returnCache(cuCache_float_data);
  if(cuCache_complex_data) memMngr.returnCache(cuCache_complex_data);
}

__device__ Real cugetVal(mode m, complexFormat &data){
  if(m==IMAG) return data.y;
  if(m==MOD) return hipCabsf(data);
  if(m==MOD2) return data.x*data.x+data.y*data.y;
  if(m==PHASE){
    return atan2(data.y,data.x)/2/M_PI+0.5;
  }
  if(m==PHASERAD){
    return atan2(data.y,data.x);
  }
  return data.x;
}
__device__ Real cugetVal(mode m, Real &data){
  if(m==MOD2) return data*data;
  return data;
}

template <typename T>
__global__ void process(void* cudaData, pixeltype* cache, mode m, bool isFrequency=0, Real decay = 1, bool islog = 0){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  int halfrow = cuda_row>>1;
  int halfcol = cuda_column>>1;
  int targetx = x;
  int targety = y;
  if(isFrequency) {
    targetx = x<halfrow?x+halfrow:(x-halfrow);
    targety = y<halfcol?y+halfcol:(y-halfcol);
  }
  T data = ((T*)cudaData)[index];
  Real target = decay*cugetVal(m,data);
  if(target < 0) target = 0;
  if(islog){
    if(target!=0)
      target = log2f(target)*cuda_rcolor/log2f(cuda_rcolor)+cuda_rcolor;
  }else target*=cuda_rcolor;
  if(target>=cuda_rcolor) {
    target=cuda_rcolor-1;
  }
  if(target!=target) {
 //   printf("ERROR: target is NAN\n");
 //   exit(0);
  }
  cache[targetx*cuda_column+targety] = floor(target);
}
__global__ void getPhase(void* cudaData, Real* cache, mode m, bool isFrequency=0, Real decay = 1){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= cuda_row || y >= cuda_column) return;
  int index = x*cuda_column + y;
  int halfrow = cuda_row>>1;
  int halfcol = cuda_column>>1;
  int targetx = x;
  int targety = y;
  if(isFrequency) {
    targetx = x<halfrow?x+halfrow:(x-halfrow);
    targety = y<halfcol?y+halfcol:(y-halfcol);
  }
  cache[targetx*cuda_column+targety] =decay*cugetVal(m,((complexFormat*)cudaData)[index]);
}

void cuPlotter::processPhaseData(void* cudaData, const mode m, bool isFrequency, Real decay){
  cudaF(getPhase)(cudaData, (Real*)cuCache_float_data, m, isFrequency, decay);
  hipMemcpy(cv_float_data, cuCache_float_data,rows*cols*sizeof(Real), hipMemcpyDeviceToHost); 
};

void cuPlotter::saveFloatData(void* cudaData){
  hipMemcpy(cv_float_data, cudaData, rows*cols*sizeof(Real), hipMemcpyDeviceToHost); 
};
void cuPlotter::saveComplexData(void* cudaData){
  hipMemcpy(cv_complex_data, cudaData, rows*cols*sizeof(complexFormat), hipMemcpyDeviceToHost); 
};
void cuPlotter::processFloatData(void* cudaData, const mode m, bool isFrequency, Real decay, bool islog){
  cudaF(process<Real>)(cudaData, cuCache_data, m, isFrequency, decay, islog);
  hipMemcpy(cv_data, cuCache_data,rows*cols*sizeof(pixeltype), hipMemcpyDeviceToHost); 
};
void cuPlotter::processComplexData(void* cudaData, const mode m, bool isFrequency, Real decay, bool islog){
  cudaF(process<complexFormat>)(cudaData, cuCache_data, m,isFrequency, decay, islog);
  hipMemcpy(cv_data, cuCache_data,rows*cols*sizeof(pixeltype), hipMemcpyDeviceToHost); 
};
